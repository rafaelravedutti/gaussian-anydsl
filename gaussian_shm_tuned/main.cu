
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_6987;
typedef struct {
    struct_Buffer_6987 e0;
    int e1;
    int e2;
} struct_filter_6986;
typedef struct {
    struct_Buffer_6987 e0;
    struct_Buffer_6987 e1;
    int e2;
    int e3;
} struct_image_6992;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_28285(struct_filter_6986, double*, struct_image_6992, double*, struct_Buffer_6987);
__global__ void lambda_28609(struct_Buffer_6987, struct_image_6992, struct_filter_6986, double*, double*, double*);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_28285(struct_filter_6986 _28288_34329, double* _28289_34330, struct_image_6992 _28290_34331, double* _28291_34332, struct_Buffer_6987 _28292_34333) {
    int  bdimx_34339;
    int pbdimx_34339;
    int  bdimy_34345;
    int pbdimy_34345;
    int  bidx_34351;
    int pbidx_34351;
    int  bidy_34357;
    int pbidy_34357;
    int  tidx_34363;
    int ptidx_34363;
    int  tidy_34369;
    int ptidy_34369;
    double*  reserve_shared_34377;
    double* preserve_shared_34377;
    double*  reserve_shared_34385;
    double* preserve_shared_34385;
    int  _34396;
    int p_34396;
    int  _34452;
    int p_34452;
    int  _34506;
    int p_34506;
    int  _34531;
    int p_34531;
    int  _34580;
    int p_34580;
    double  sum_34582;
    double psum_34582;
    int  _34536;
    int p_34536;
    int  _34457;
    int p_34457;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    bdimx_34339 = blockDim_x();
    pbdimx_34339 = bdimx_34339;
    l34337: ;
        bdimx_34339 = pbdimx_34339;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bdimy_34345 = blockDim_y();
        pbdimy_34345 = bdimy_34345;
    l34343: ;
        bdimy_34345 = pbdimy_34345;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bidx_34351 = blockIdx_x();
        pbidx_34351 = bidx_34351;
    l34349: ;
        bidx_34351 = pbidx_34351;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bidy_34357 = blockIdx_y();
        pbidy_34357 = bidy_34357;
    l34355: ;
        bidy_34357 = pbidy_34357;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        tidx_34363 = threadIdx_x();
        ptidx_34363 = tidx_34363;
    l34361: ;
        tidx_34363 = ptidx_34363;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        tidy_34369 = threadIdx_y();
        ptidy_34369 = tidy_34369;
    l34367: ;
        tidy_34369 = ptidy_34369;
        #line 215 "impala/gpu_device_shm.impala"
        __shared__ double reserver_reserve_shared_34377[938];
        preserve_shared_34377 = reserver_reserve_shared_34377;
    l34375: ;
        reserve_shared_34377 = preserve_shared_34377;
        #line 223 "impala/gpu_device_shm.impala"
        __shared__ double reserver_reserve_shared_34385[49];
        preserve_shared_34385 = reserver_reserve_shared_34385;
    l34383: ;
        reserve_shared_34385 = preserve_shared_34385;
        #line 203 "impala/gpu_device_shm.impala"
        int _34419;
        _34419 = bidy_34357 * bdimy_34345;
        #line 201 "impala/gpu_device_shm.impala"
        int _34410;
        _34410 = bidx_34351 * bdimx_34339;
        #line 201 "impala/gpu_device_shm.impala"
        int gid_x_34411;
        gid_x_34411 = _34410 + tidx_34363;
        #line 11 "impala/main.impala"
        int _34425;
        _34425 = _28290_34331.e3;
        #line 11 "impala/main.impala"
        int _34416;
        _34416 = _28290_34331.e2;
        #line 4 "impala/gaussian.impala"
        int _34399;
        _34399 = _28288_34329.e1;
        #line 203 "impala/gpu_device_shm.impala"
        int gid_y_34420;
        gid_y_34420 = _34419 + tidy_34369;
        #line 207 "impala/gpu_device_shm.impala"
        int _34389;
        _34389 = _28288_34329.e2;
        #line 4 "impala/gaussian.impala"
        int h_anchor_34400;
        h_anchor_34400 = _34399 / 2;
        #line 207 "impala/gpu_device_shm.impala"
        int extend_height_34390;
        extend_height_34390 = _34389 / 2;
        #line 209 "impala/gpu_device_shm.impala"
        int _34401;
        _34401 = 2 * h_anchor_34400;
        #line 211 "impala/gpu_device_shm.impala"
        int _34391;
        _34391 = 2 * extend_height_34390;
        #line 209 "impala/gpu_device_shm.impala"
        int shm_dimx_34402;
        shm_dimx_34402 = bdimx_34339 + _34401;
        #line 211 "impala/gpu_device_shm.impala"
        int shm_dimy_34392;
        shm_dimy_34392 = bdimy_34345 + _34391;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34393;
        _34393 = 0 < shm_dimy_34392;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34393) goto l34394; else goto l34629;
    l34629: ;
        #line 253 "impala/gpu_device_shm.impala"
        goto l34501;
    l34394: ;
        #line 241 "impala/gpu_device_shm.impala"
        int _34412;
        _34412 = gid_x_34411 - h_anchor_34400;
        #line 239 "impala/gpu_device_shm.impala"
        bool _34408;
        _34408 = tidy_34369 < shm_dimy_34392;
        #line 243 "impala/gpu_device_shm.impala"
        int _34421;
        _34421 = gid_y_34420 - extend_height_34390;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34422;
        _34422 = 0 <= _34421;
        #line 248 "impala/gpu_device_shm.impala"
        int _34437;
        _34437 = tidy_34369 * shm_dimx_34402;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34426;
        _34426 = _34421 < _34425;
        #line 249 "impala/gpu_device_shm.impala"
        int _34431;
        _34431 = _34421 * _34416;
        #line 52 "impala/gpu_device_shm.impala"
        p_34396 = 0;
        goto l34395;
    l34395: ;
        _34396 = p_34396;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34403;
        _34403 = _34396 < shm_dimx_34402;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34403) goto l34404; else goto l34450;
    l34450: ;
        #line 52 "impala/gpu_device_shm.impala"
        p_34452 = bdimy_34345;
        goto l34451;
    l34451: ;
        _34452 = p_34452;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34454;
        _34454 = _34452 < shm_dimy_34392;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34454) goto l34455; else goto l34500;
    l34500: ;
        #line 253 "impala/gpu_device_shm.impala"
        goto l34501;
    l34501: ;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34503;
        _34503 = 0 < _34389;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34503) goto l34504; else goto l34628;
    l34628: ;
        #line 271 "impala/gpu_device_shm.impala"
        goto l34563;
    l34504: ;
        #line 265 "impala/gpu_device_shm.impala"
        bool _34513;
        _34513 = tidy_34369 < _34389;
        #line 267 "impala/gpu_device_shm.impala"
        int _34518;
        _34518 = tidy_34369 * _34399;
        #line 52 "impala/gpu_device_shm.impala"
        p_34506 = 0;
        goto l34505;
    l34505: ;
        _34506 = p_34506;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34508;
        _34508 = _34506 < _34399;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34508) goto l34509; else goto l34529;
    l34529: ;
        #line 52 "impala/gpu_device_shm.impala"
        p_34531 = bdimy_34345;
        goto l34530;
    l34530: ;
        _34531 = p_34531;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34533;
        _34533 = _34531 < _34389;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34533) goto l34534; else goto l34562;
    l34562: ;
        #line 271 "impala/gpu_device_shm.impala"
        goto l34563;
    l34563: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        __syncthreads();
    l34568: ;
        #line 277 "impala/gpu_device_shm.impala"
        bool _34570;
        _34570 = gid_x_34411 < _34416;
        #line 277 "impala/gpu_device_shm.impala"
        if (_34570) goto l34571; else goto l34627;
    l34627: ;
        #line 280 "impala/gpu_device_shm.impala"
        goto l34626;
    l34571: ;
        #line 277 "impala/gpu_device_shm.impala"
        bool _34572;
        _34572 = gid_y_34420 < _34425;
        #line 277 "impala/gpu_device_shm.impala"
        if (_34572) goto l34573; else goto l34625;
    l34625: ;
        #line 280 "impala/gpu_device_shm.impala"
        goto l34626;
    l34626: ;
        return ;
    l34573: ;
        #line 78 "impala/gpu_device_shm.impala"
        char* _34607;
        _34607 = _28292_34333.e1;
        #line 78 "impala/gpu_device_shm.impala"
        double* _34608;
        union { double* dst; char* src; } u_34608;
        u_34608.src = _34607;
        _34608 = u_34608.dst;
        #line 217 "impala/gpu_device_shm.impala"
        int _34597;
        _34597 = h_anchor_34400 - _34410;
        #line 78 "impala/gpu_device_shm.impala"
        int _34609;
        _34609 = gid_y_34420 * _34416;
        #line 218 "impala/gpu_device_shm.impala"
        int _34592;
        _34592 = extend_height_34390 - _34419;
        #line 86 "impala/gpu_device_shm.impala"
        int _34593;
        _34593 = gid_y_34420 + _34592;
        #line 78 "impala/gpu_device_shm.impala"
        int _34610;
        _34610 = _34609 + gid_x_34411;
        #line 17 "impala/gaussian.impala"
        bool _34574;
        _34574 = h_anchor_34400 <= gid_x_34411;
        #line 78 "impala/gpu_device_shm.impala"
        double* _34611;
        _34611 = _34608 + _34610;
        #line 86 "impala/gpu_device_shm.impala"
        int _34594;
        _34594 = _34593 * shm_dimx_34402;
        #line 17 "impala/gaussian.impala"
        if (_34574) goto l34575; else goto l34624;
    l34624: ;
        #line 27 "impala/gaussian.impala"
        goto l34616;
    l34575: ;
        #line 17 "impala/gaussian.impala"
        int _34576;
        _34576 = _34416 - h_anchor_34400;
        #line 17 "impala/gaussian.impala"
        bool _34577;
        _34577 = gid_x_34411 < _34576;
        #line 17 "impala/gaussian.impala"
        if (_34577) goto l34578; else goto l34615;
    l34615: ;
        #line 27 "impala/gaussian.impala"
        goto l34616;
    l34616: ;
        #line 86 "impala/gpu_device_shm.impala"
        int _34617;
        _34617 = _34594 + gid_x_34411;
        #line 86 "impala/gpu_device_shm.impala"
        int _34618;
        _34618 = _34617 + _34597;
        #line 86 "impala/gpu_device_shm.impala"
        double* _34619;
        _34619 = reserve_shared_34377 + _34618;
        #line 86 "impala/gpu_device_shm.impala"
        double _34620;
        _34620 = *_34619;
        #line 86 "impala/gpu_device_shm.impala"
        double _34622;
        _34622 = _34620;
        #line 78 "impala/gpu_device_shm.impala"
        *_34611 = _34622;
        return ;
    l34578: ;
        #line 19 "impala/gaussian.impala"
        int _34584;
        _34584 = 1 + h_anchor_34400;
        #line 19 "impala/gaussian.impala"
        int _34613;
        _34613 = 0 - h_anchor_34400;
        #line 27 "impala/gpu_device_shm.impala"
        p_34580 = _34613;
        psum_34582 = 0.000000e+00;
        goto l34579;
    l34579: ;
        _34580 = p_34580;
        sum_34582 = psum_34582;
        #line 27 "impala/gpu_device_shm.impala"
        bool _34585;
        _34585 = _34580 < _34584;
        #line 27 "impala/gpu_device_shm.impala"
        if (_34585) goto l34586; else goto l34606;
    l34606: ;
        #line 78 "impala/gpu_device_shm.impala"
        *_34611 = sum_34582;
        return ;
    l34586: ;
        #line 31 "impala/gpu_device_shm.impala"
        int _34587;
        _34587 = 1 + _34580;
        #line 21 "impala/gaussian.impala"
        int _34595;
        _34595 = gid_x_34411 + _34580;
        #line 21 "impala/gaussian.impala"
        int _34588;
        _34588 = _34580 + h_anchor_34400;
        #line 86 "impala/gpu_device_shm.impala"
        int _34596;
        _34596 = _34594 + _34595;
        #line 90 "impala/gpu_device_shm.impala"
        double* i_34589;
        i_34589 = reserve_shared_34385 + _34588;
        #line 86 "impala/gpu_device_shm.impala"
        int _34598;
        _34598 = _34596 + _34597;
        #line 91 "impala/gpu_device_shm.impala"
        double _34590;
        _34590 = *i_34589;
        #line 86 "impala/gpu_device_shm.impala"
        double* _34599;
        _34599 = reserve_shared_34377 + _34598;
        #line 91 "impala/gpu_device_shm.impala"
        double _34602;
        _34602 = _34590;
        #line 86 "impala/gpu_device_shm.impala"
        double _34600;
        _34600 = *_34599;
        #line 86 "impala/gpu_device_shm.impala"
        double _34603;
        _34603 = _34600;
        #line 21 "impala/gaussian.impala"
        double _34604;
        _34604 = _34602 * _34603;
        #line 21 "impala/gaussian.impala"
        double _34605;
        _34605 = sum_34582 + _34604;
        #line 27 "impala/gpu_device_shm.impala"
        p_34580 = _34587;
        psum_34582 = _34605;
        goto l34579;
    l34534: ;
        #line 262 "impala/gpu_device_shm.impala"
        int shm_index_y_34543;
        shm_index_y_34543 = tidy_34369 + _34531;
        #line 265 "impala/gpu_device_shm.impala"
        bool _34544;
        _34544 = shm_index_y_34543 < _34389;
        #line 267 "impala/gpu_device_shm.impala"
        int _34549;
        _34549 = shm_index_y_34543 * _34399;
        #line 52 "impala/gpu_device_shm.impala"
        p_34536 = 0;
        goto l34535;
    l34535: ;
        _34536 = p_34536;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34538;
        _34538 = _34536 < _34399;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34538) goto l34539; else goto l34560;
    l34560: ;
        #line 56 "impala/gpu_device_shm.impala"
        int _34561;
        _34561 = _34531 + bdimy_34345;
        #line 52 "impala/gpu_device_shm.impala"
        p_34531 = _34561;
        goto l34530;
    l34539: ;
        #line 260 "impala/gpu_device_shm.impala"
        int shm_index_x_34540;
        shm_index_x_34540 = tidx_34363 + _34536;
        #line 265 "impala/gpu_device_shm.impala"
        bool _34541;
        _34541 = shm_index_x_34540 < _34399;
        #line 265 "impala/gpu_device_shm.impala"
        if (_34541) goto l34542; else goto l34559;
    l34559: ;
        #line 269 "impala/gpu_device_shm.impala"
        goto l34558;
    l34542: ;
        #line 265 "impala/gpu_device_shm.impala"
        if (_34544) goto l34545; else goto l34557;
    l34557: ;
        #line 269 "impala/gpu_device_shm.impala"
        goto l34558;
    l34558: ;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34546;
    l34545: ;
        #line 267 "impala/gpu_device_shm.impala"
        int _34550;
        _34550 = _34549 + shm_index_x_34540;
        #line 267 "impala/gpu_device_shm.impala"
        double* _34551;
        _34551 = _28291_34332 + _34550;
        #line 267 "impala/gpu_device_shm.impala"
        double* _34554;
        _34554 = reserve_shared_34385 + _34550;
        #line 268 "impala/gpu_device_shm.impala"
        double _34552;
        _34552 = *_34551;
        #line 268 "impala/gpu_device_shm.impala"
        double _34555;
        _34555 = _34552;
        #line 267 "impala/gpu_device_shm.impala"
        *_34554 = _34555;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34546;
    l34546: ;
        #line 56 "impala/gpu_device_shm.impala"
        int _34548;
        _34548 = _34536 + bdimx_34339;
        #line 52 "impala/gpu_device_shm.impala"
        p_34536 = _34548;
        goto l34535;
    l34509: ;
        #line 260 "impala/gpu_device_shm.impala"
        int shm_index_x_34510;
        shm_index_x_34510 = tidx_34363 + _34506;
        #line 265 "impala/gpu_device_shm.impala"
        bool _34511;
        _34511 = shm_index_x_34510 < _34399;
        #line 265 "impala/gpu_device_shm.impala"
        if (_34511) goto l34512; else goto l34528;
    l34528: ;
        #line 269 "impala/gpu_device_shm.impala"
        goto l34527;
    l34512: ;
        #line 265 "impala/gpu_device_shm.impala"
        if (_34513) goto l34514; else goto l34526;
    l34526: ;
        #line 269 "impala/gpu_device_shm.impala"
        goto l34527;
    l34527: ;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34515;
    l34514: ;
        #line 267 "impala/gpu_device_shm.impala"
        int _34519;
        _34519 = _34518 + shm_index_x_34510;
        #line 267 "impala/gpu_device_shm.impala"
        double* _34523;
        _34523 = reserve_shared_34385 + _34519;
        #line 267 "impala/gpu_device_shm.impala"
        double* _34520;
        _34520 = _28291_34332 + _34519;
        #line 268 "impala/gpu_device_shm.impala"
        double _34521;
        _34521 = *_34520;
        #line 268 "impala/gpu_device_shm.impala"
        double _34524;
        _34524 = _34521;
        #line 267 "impala/gpu_device_shm.impala"
        *_34523 = _34524;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34515;
    l34515: ;
        #line 56 "impala/gpu_device_shm.impala"
        int _34517;
        _34517 = _34506 + bdimx_34339;
        #line 52 "impala/gpu_device_shm.impala"
        p_34506 = _34517;
        goto l34505;
    l34455: ;
        #line 243 "impala/gpu_device_shm.impala"
        int img_index_y_34472;
        img_index_y_34472 = _34421 + _34452;
        #line 249 "impala/gpu_device_shm.impala"
        int _34480;
        _34480 = img_index_y_34472 * _34416;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34475;
        _34475 = img_index_y_34472 < _34425;
        #line 236 "impala/gpu_device_shm.impala"
        int shm_index_y_34464;
        shm_index_y_34464 = tidy_34369 + _34452;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34473;
        _34473 = 0 <= img_index_y_34472;
        #line 239 "impala/gpu_device_shm.impala"
        bool _34465;
        _34465 = shm_index_y_34464 < shm_dimy_34392;
        #line 248 "impala/gpu_device_shm.impala"
        int _34485;
        _34485 = shm_index_y_34464 * shm_dimx_34402;
        #line 52 "impala/gpu_device_shm.impala"
        p_34457 = 0;
        goto l34456;
    l34456: ;
        _34457 = p_34457;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34459;
        _34459 = _34457 < shm_dimx_34402;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34459) goto l34460; else goto l34498;
    l34498: ;
        #line 56 "impala/gpu_device_shm.impala"
        int _34499;
        _34499 = _34452 + bdimy_34345;
        #line 52 "impala/gpu_device_shm.impala"
        p_34452 = _34499;
        goto l34451;
    l34460: ;
        #line 234 "impala/gpu_device_shm.impala"
        int shm_index_x_34461;
        shm_index_x_34461 = tidx_34363 + _34457;
        #line 239 "impala/gpu_device_shm.impala"
        bool _34462;
        _34462 = shm_index_x_34461 < shm_dimx_34402;
        #line 239 "impala/gpu_device_shm.impala"
        if (_34462) goto l34463; else goto l34497;
    l34497: ;
        #line 251 "impala/gpu_device_shm.impala"
        goto l34496;
    l34463: ;
        #line 239 "impala/gpu_device_shm.impala"
        if (_34465) goto l34466; else goto l34495;
    l34495: ;
        #line 251 "impala/gpu_device_shm.impala"
        goto l34496;
    l34496: ;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34477;
    l34466: ;
        #line 241 "impala/gpu_device_shm.impala"
        int img_index_x_34467;
        img_index_x_34467 = _34412 + _34457;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34468;
        _34468 = 0 <= img_index_x_34467;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34468) goto l34469; else goto l34494;
    l34494: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34491;
    l34469: ;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34470;
        _34470 = img_index_x_34467 < _34416;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34470) goto l34471; else goto l34493;
    l34493: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34491;
    l34471: ;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34473) goto l34474; else goto l34492;
    l34492: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34491;
    l34474: ;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34475) goto l34476; else goto l34490;
    l34490: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34491;
    l34491: ;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34477;
    l34476: ;
        #line 249 "impala/gpu_device_shm.impala"
        int _34481;
        _34481 = _34480 + img_index_x_34467;
        #line 248 "impala/gpu_device_shm.impala"
        int _34486;
        _34486 = _34485 + shm_index_x_34461;
        #line 249 "impala/gpu_device_shm.impala"
        double* _34482;
        _34482 = _28289_34330 + _34481;
        #line 248 "impala/gpu_device_shm.impala"
        double* _34487;
        _34487 = reserve_shared_34377 + _34486;
        #line 249 "impala/gpu_device_shm.impala"
        double _34483;
        _34483 = *_34482;
        #line 249 "impala/gpu_device_shm.impala"
        double _34488;
        _34488 = _34483;
        #line 248 "impala/gpu_device_shm.impala"
        *_34487 = _34488;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34477;
    l34477: ;
        #line 56 "impala/gpu_device_shm.impala"
        int _34479;
        _34479 = _34457 + bdimx_34339;
        #line 52 "impala/gpu_device_shm.impala"
        p_34457 = _34479;
        goto l34456;
    l34404: ;
        #line 234 "impala/gpu_device_shm.impala"
        int shm_index_x_34405;
        shm_index_x_34405 = tidx_34363 + _34396;
        #line 239 "impala/gpu_device_shm.impala"
        bool _34406;
        _34406 = shm_index_x_34405 < shm_dimx_34402;
        #line 239 "impala/gpu_device_shm.impala"
        if (_34406) goto l34407; else goto l34449;
    l34449: ;
        #line 251 "impala/gpu_device_shm.impala"
        goto l34448;
    l34407: ;
        #line 239 "impala/gpu_device_shm.impala"
        if (_34408) goto l34409; else goto l34447;
    l34447: ;
        #line 251 "impala/gpu_device_shm.impala"
        goto l34448;
    l34448: ;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34428;
    l34409: ;
        #line 241 "impala/gpu_device_shm.impala"
        int img_index_x_34413;
        img_index_x_34413 = _34412 + _34396;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34414;
        _34414 = 0 <= img_index_x_34413;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34414) goto l34415; else goto l34446;
    l34446: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34443;
    l34415: ;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34417;
        _34417 = img_index_x_34413 < _34416;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34417) goto l34418; else goto l34445;
    l34445: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34443;
    l34418: ;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34422) goto l34423; else goto l34444;
    l34444: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34443;
    l34423: ;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34426) goto l34427; else goto l34442;
    l34442: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34443;
    l34443: ;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34428;
    l34427: ;
        #line 249 "impala/gpu_device_shm.impala"
        int _34432;
        _34432 = _34431 + img_index_x_34413;
        #line 248 "impala/gpu_device_shm.impala"
        int _34438;
        _34438 = _34437 + shm_index_x_34405;
        #line 248 "impala/gpu_device_shm.impala"
        double* _34439;
        _34439 = reserve_shared_34377 + _34438;
        #line 249 "impala/gpu_device_shm.impala"
        double* _34433;
        _34433 = _28289_34330 + _34432;
        #line 249 "impala/gpu_device_shm.impala"
        double _34434;
        _34434 = *_34433;
        #line 249 "impala/gpu_device_shm.impala"
        double _34440;
        _34440 = _34434;
        #line 248 "impala/gpu_device_shm.impala"
        *_34439 = _34440;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34428;
    l34428: ;
        #line 56 "impala/gpu_device_shm.impala"
        int _34430;
        _34430 = _34396 + bdimx_34339;
        #line 52 "impala/gpu_device_shm.impala"
        p_34396 = _34430;
        goto l34395;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_28609(struct_Buffer_6987 _28612_34633, struct_image_6992 _28613_34634, struct_filter_6986 _28614_34635, double* _28615_34636, double* _28616_34637, double* _28617_34638) {
    int  bdimx_34641;
    int pbdimx_34641;
    int  bdimy_34644;
    int pbdimy_34644;
    int  bidx_34647;
    int pbidx_34647;
    int  bidy_34650;
    int pbidy_34650;
    int  tidx_34653;
    int ptidx_34653;
    int  tidy_34656;
    int ptidy_34656;
    double*  reserve_shared_34659;
    double* preserve_shared_34659;
    double*  reserve_shared_34662;
    double* preserve_shared_34662;
    int  _34670;
    int p_34670;
    int  _34723;
    int p_34723;
    int  _34777;
    int p_34777;
    int  _34802;
    int p_34802;
    int  _34848;
    int p_34848;
    double  sum_34850;
    double psum_34850;
    int  _34807;
    int p_34807;
    int  _34728;
    int p_34728;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    bdimx_34641 = blockDim_x();
    pbdimx_34641 = bdimx_34641;
    l34639: ;
        bdimx_34641 = pbdimx_34641;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bdimy_34644 = blockDim_y();
        pbdimy_34644 = bdimy_34644;
    l34642: ;
        bdimy_34644 = pbdimy_34644;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bidx_34647 = blockIdx_x();
        pbidx_34647 = bidx_34647;
    l34645: ;
        bidx_34647 = pbidx_34647;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bidy_34650 = blockIdx_y();
        pbidy_34650 = bidy_34650;
    l34648: ;
        bidy_34650 = pbidy_34650;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        tidx_34653 = threadIdx_x();
        ptidx_34653 = tidx_34653;
    l34651: ;
        tidx_34653 = ptidx_34653;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        tidy_34656 = threadIdx_y();
        ptidy_34656 = tidy_34656;
    l34654: ;
        tidy_34656 = ptidy_34656;
        #line 215 "impala/gpu_device_shm.impala"
        __shared__ double reserver_reserve_shared_34659[938];
        preserve_shared_34659 = reserver_reserve_shared_34659;
    l34657: ;
        reserve_shared_34659 = preserve_shared_34659;
        #line 223 "impala/gpu_device_shm.impala"
        __shared__ double reserver_reserve_shared_34662[49];
        preserve_shared_34662 = reserver_reserve_shared_34662;
    l34660: ;
        reserve_shared_34662 = preserve_shared_34662;
        #line 11 "impala/main.impala"
        int _34697;
        _34697 = _28613_34634.e3;
        #line 6 "impala/gaussian.impala"
        int _34663;
        _34663 = _28614_34635.e2;
        #line 201 "impala/gpu_device_shm.impala"
        int _34683;
        _34683 = bidx_34647 * bdimx_34641;
        #line 201 "impala/gpu_device_shm.impala"
        int gid_x_34684;
        gid_x_34684 = _34683 + tidx_34653;
        #line 203 "impala/gpu_device_shm.impala"
        int _34692;
        _34692 = bidy_34650 * bdimy_34644;
        #line 11 "impala/main.impala"
        int _34689;
        _34689 = _28613_34634.e2;
        #line 205 "impala/gpu_device_shm.impala"
        int _34672;
        _34672 = _28614_34635.e1;
        #line 6 "impala/gaussian.impala"
        int v_anchor_34664;
        v_anchor_34664 = _34663 / 2;
        #line 203 "impala/gpu_device_shm.impala"
        int gid_y_34693;
        gid_y_34693 = _34692 + tidy_34656;
        #line 205 "impala/gpu_device_shm.impala"
        int extend_width_34673;
        extend_width_34673 = _34672 / 2;
        #line 211 "impala/gpu_device_shm.impala"
        int _34665;
        _34665 = 2 * v_anchor_34664;
        #line 209 "impala/gpu_device_shm.impala"
        int _34674;
        _34674 = 2 * extend_width_34673;
        #line 211 "impala/gpu_device_shm.impala"
        int shm_dimy_34666;
        shm_dimy_34666 = bdimy_34644 + _34665;
        #line 209 "impala/gpu_device_shm.impala"
        int shm_dimx_34675;
        shm_dimx_34675 = bdimx_34641 + _34674;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34667;
        _34667 = 0 < shm_dimy_34666;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34667) goto l34668; else goto l34897;
    l34897: ;
        #line 253 "impala/gpu_device_shm.impala"
        goto l34772;
    l34668: ;
        #line 248 "impala/gpu_device_shm.impala"
        int _34708;
        _34708 = tidy_34656 * shm_dimx_34675;
        #line 239 "impala/gpu_device_shm.impala"
        bool _34681;
        _34681 = tidy_34656 < shm_dimy_34666;
        #line 243 "impala/gpu_device_shm.impala"
        int _34694;
        _34694 = gid_y_34693 - v_anchor_34664;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34695;
        _34695 = 0 <= _34694;
        #line 241 "impala/gpu_device_shm.impala"
        int _34685;
        _34685 = gid_x_34684 - extend_width_34673;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34698;
        _34698 = _34694 < _34697;
        #line 249 "impala/gpu_device_shm.impala"
        int _34703;
        _34703 = _34694 * _34689;
        #line 52 "impala/gpu_device_shm.impala"
        p_34670 = 0;
        goto l34669;
    l34669: ;
        _34670 = p_34670;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34676;
        _34676 = _34670 < shm_dimx_34675;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34676) goto l34677; else goto l34721;
    l34721: ;
        #line 52 "impala/gpu_device_shm.impala"
        p_34723 = bdimy_34644;
        goto l34722;
    l34722: ;
        _34723 = p_34723;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34725;
        _34725 = _34723 < shm_dimy_34666;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34725) goto l34726; else goto l34771;
    l34771: ;
        #line 253 "impala/gpu_device_shm.impala"
        goto l34772;
    l34772: ;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34774;
        _34774 = 0 < _34663;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34774) goto l34775; else goto l34896;
    l34896: ;
        #line 271 "impala/gpu_device_shm.impala"
        goto l34834;
    l34775: ;
        #line 265 "impala/gpu_device_shm.impala"
        bool _34784;
        _34784 = tidy_34656 < _34663;
        #line 267 "impala/gpu_device_shm.impala"
        int _34789;
        _34789 = tidy_34656 * _34672;
        #line 52 "impala/gpu_device_shm.impala"
        p_34777 = 0;
        goto l34776;
    l34776: ;
        _34777 = p_34777;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34779;
        _34779 = _34777 < _34672;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34779) goto l34780; else goto l34800;
    l34800: ;
        #line 52 "impala/gpu_device_shm.impala"
        p_34802 = bdimy_34644;
        goto l34801;
    l34801: ;
        _34802 = p_34802;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34804;
        _34804 = _34802 < _34663;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34804) goto l34805; else goto l34833;
    l34833: ;
        #line 271 "impala/gpu_device_shm.impala"
        goto l34834;
    l34834: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        __syncthreads();
    l34836: ;
        #line 277 "impala/gpu_device_shm.impala"
        bool _34838;
        _34838 = gid_x_34684 < _34689;
        #line 277 "impala/gpu_device_shm.impala"
        if (_34838) goto l34839; else goto l34895;
    l34895: ;
        #line 280 "impala/gpu_device_shm.impala"
        goto l34894;
    l34839: ;
        #line 277 "impala/gpu_device_shm.impala"
        bool _34840;
        _34840 = gid_y_34693 < _34697;
        #line 277 "impala/gpu_device_shm.impala"
        if (_34840) goto l34841; else goto l34893;
    l34893: ;
        #line 280 "impala/gpu_device_shm.impala"
        goto l34894;
    l34894: ;
        return ;
    l34841: ;
        #line 39 "impala/gaussian.impala"
        bool _34842;
        _34842 = v_anchor_34664 <= gid_y_34693;
        #line 217 "impala/gpu_device_shm.impala"
        int _34864;
        _34864 = extend_width_34673 - _34683;
        #line 218 "impala/gpu_device_shm.impala"
        int _34860;
        _34860 = v_anchor_34664 - _34692;
        #line 78 "impala/gpu_device_shm.impala"
        char* _34874;
        _34874 = _28612_34633.e1;
        #line 78 "impala/gpu_device_shm.impala"
        double* _34875;
        union { double* dst; char* src; } u_34875;
        u_34875.src = _34874;
        _34875 = u_34875.dst;
        #line 78 "impala/gpu_device_shm.impala"
        int _34876;
        _34876 = gid_y_34693 * _34689;
        #line 78 "impala/gpu_device_shm.impala"
        int _34877;
        _34877 = _34876 + gid_x_34684;
        #line 78 "impala/gpu_device_shm.impala"
        double* _34878;
        _34878 = _34875 + _34877;
        #line 39 "impala/gaussian.impala"
        if (_34842) goto l34843; else goto l34892;
    l34892: ;
        #line 49 "impala/gaussian.impala"
        goto l34882;
    l34843: ;
        #line 39 "impala/gaussian.impala"
        int _34844;
        _34844 = _34697 - v_anchor_34664;
        #line 39 "impala/gaussian.impala"
        bool _34845;
        _34845 = gid_y_34693 < _34844;
        #line 39 "impala/gaussian.impala"
        if (_34845) goto l34846; else goto l34881;
    l34881: ;
        #line 49 "impala/gaussian.impala"
        goto l34882;
    l34882: ;
        #line 86 "impala/gpu_device_shm.impala"
        int _34883;
        _34883 = gid_y_34693 + _34860;
        #line 86 "impala/gpu_device_shm.impala"
        int _34884;
        _34884 = _34883 * shm_dimx_34675;
        #line 86 "impala/gpu_device_shm.impala"
        int _34885;
        _34885 = _34884 + gid_x_34684;
        #line 86 "impala/gpu_device_shm.impala"
        int _34886;
        _34886 = _34885 + _34864;
        #line 86 "impala/gpu_device_shm.impala"
        double* _34887;
        _34887 = reserve_shared_34659 + _34886;
        #line 86 "impala/gpu_device_shm.impala"
        double _34888;
        _34888 = *_34887;
        #line 86 "impala/gpu_device_shm.impala"
        double _34890;
        _34890 = _34888;
        #line 78 "impala/gpu_device_shm.impala"
        *_34878 = _34890;
        return ;
    l34846: ;
        #line 41 "impala/gaussian.impala"
        int _34851;
        _34851 = 1 + v_anchor_34664;
        #line 41 "impala/gaussian.impala"
        int _34880;
        _34880 = 0 - v_anchor_34664;
        #line 27 "impala/gpu_device_shm.impala"
        p_34848 = _34880;
        psum_34850 = 0.000000e+00;
        goto l34847;
    l34847: ;
        _34848 = p_34848;
        sum_34850 = psum_34850;
        #line 27 "impala/gpu_device_shm.impala"
        bool _34852;
        _34852 = _34848 < _34851;
        #line 27 "impala/gpu_device_shm.impala"
        if (_34852) goto l34853; else goto l34873;
    l34873: ;
        #line 78 "impala/gpu_device_shm.impala"
        *_34878 = sum_34850;
        return ;
    l34853: ;
        #line 43 "impala/gaussian.impala"
        int _34859;
        _34859 = gid_y_34693 + _34848;
        #line 31 "impala/gpu_device_shm.impala"
        int _34854;
        _34854 = 1 + _34848;
        #line 43 "impala/gaussian.impala"
        int _34855;
        _34855 = _34848 + v_anchor_34664;
        #line 90 "impala/gpu_device_shm.impala"
        double* i_34856;
        i_34856 = reserve_shared_34662 + _34855;
        #line 86 "impala/gpu_device_shm.impala"
        int _34861;
        _34861 = _34859 + _34860;
        #line 91 "impala/gpu_device_shm.impala"
        double _34857;
        _34857 = *i_34856;
        #line 86 "impala/gpu_device_shm.impala"
        int _34862;
        _34862 = _34861 * shm_dimx_34675;
        #line 91 "impala/gpu_device_shm.impala"
        double _34869;
        _34869 = _34857;
        #line 86 "impala/gpu_device_shm.impala"
        int _34863;
        _34863 = _34862 + gid_x_34684;
        #line 86 "impala/gpu_device_shm.impala"
        int _34865;
        _34865 = _34863 + _34864;
        #line 86 "impala/gpu_device_shm.impala"
        double* _34866;
        _34866 = reserve_shared_34659 + _34865;
        #line 86 "impala/gpu_device_shm.impala"
        double _34867;
        _34867 = *_34866;
        #line 86 "impala/gpu_device_shm.impala"
        double _34870;
        _34870 = _34867;
        #line 43 "impala/gaussian.impala"
        double _34871;
        _34871 = _34869 * _34870;
        #line 43 "impala/gaussian.impala"
        double _34872;
        _34872 = sum_34850 + _34871;
        #line 27 "impala/gpu_device_shm.impala"
        p_34848 = _34854;
        psum_34850 = _34872;
        goto l34847;
    l34805: ;
        #line 262 "impala/gpu_device_shm.impala"
        int shm_index_y_34814;
        shm_index_y_34814 = tidy_34656 + _34802;
        #line 265 "impala/gpu_device_shm.impala"
        bool _34815;
        _34815 = shm_index_y_34814 < _34663;
        #line 267 "impala/gpu_device_shm.impala"
        int _34820;
        _34820 = shm_index_y_34814 * _34672;
        #line 52 "impala/gpu_device_shm.impala"
        p_34807 = 0;
        goto l34806;
    l34806: ;
        _34807 = p_34807;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34809;
        _34809 = _34807 < _34672;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34809) goto l34810; else goto l34831;
    l34831: ;
        #line 56 "impala/gpu_device_shm.impala"
        int _34832;
        _34832 = _34802 + bdimy_34644;
        #line 52 "impala/gpu_device_shm.impala"
        p_34802 = _34832;
        goto l34801;
    l34810: ;
        #line 260 "impala/gpu_device_shm.impala"
        int shm_index_x_34811;
        shm_index_x_34811 = tidx_34653 + _34807;
        #line 265 "impala/gpu_device_shm.impala"
        bool _34812;
        _34812 = shm_index_x_34811 < _34672;
        #line 265 "impala/gpu_device_shm.impala"
        if (_34812) goto l34813; else goto l34830;
    l34830: ;
        #line 269 "impala/gpu_device_shm.impala"
        goto l34829;
    l34813: ;
        #line 265 "impala/gpu_device_shm.impala"
        if (_34815) goto l34816; else goto l34828;
    l34828: ;
        #line 269 "impala/gpu_device_shm.impala"
        goto l34829;
    l34829: ;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34817;
    l34816: ;
        #line 267 "impala/gpu_device_shm.impala"
        int _34821;
        _34821 = _34820 + shm_index_x_34811;
        #line 267 "impala/gpu_device_shm.impala"
        double* _34825;
        _34825 = reserve_shared_34662 + _34821;
        #line 267 "impala/gpu_device_shm.impala"
        double* _34822;
        _34822 = _28615_34636 + _34821;
        #line 268 "impala/gpu_device_shm.impala"
        double _34823;
        _34823 = *_34822;
        #line 268 "impala/gpu_device_shm.impala"
        double _34826;
        _34826 = _34823;
        #line 267 "impala/gpu_device_shm.impala"
        *_34825 = _34826;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34817;
    l34817: ;
        #line 56 "impala/gpu_device_shm.impala"
        int _34819;
        _34819 = _34807 + bdimx_34641;
        #line 52 "impala/gpu_device_shm.impala"
        p_34807 = _34819;
        goto l34806;
    l34780: ;
        #line 260 "impala/gpu_device_shm.impala"
        int shm_index_x_34781;
        shm_index_x_34781 = tidx_34653 + _34777;
        #line 265 "impala/gpu_device_shm.impala"
        bool _34782;
        _34782 = shm_index_x_34781 < _34672;
        #line 265 "impala/gpu_device_shm.impala"
        if (_34782) goto l34783; else goto l34799;
    l34799: ;
        #line 269 "impala/gpu_device_shm.impala"
        goto l34798;
    l34783: ;
        #line 265 "impala/gpu_device_shm.impala"
        if (_34784) goto l34785; else goto l34797;
    l34797: ;
        #line 269 "impala/gpu_device_shm.impala"
        goto l34798;
    l34798: ;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34786;
    l34785: ;
        #line 267 "impala/gpu_device_shm.impala"
        int _34790;
        _34790 = _34789 + shm_index_x_34781;
        #line 267 "impala/gpu_device_shm.impala"
        double* _34794;
        _34794 = reserve_shared_34662 + _34790;
        #line 267 "impala/gpu_device_shm.impala"
        double* _34791;
        _34791 = _28615_34636 + _34790;
        #line 268 "impala/gpu_device_shm.impala"
        double _34792;
        _34792 = *_34791;
        #line 268 "impala/gpu_device_shm.impala"
        double _34795;
        _34795 = _34792;
        #line 267 "impala/gpu_device_shm.impala"
        *_34794 = _34795;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34786;
    l34786: ;
        #line 56 "impala/gpu_device_shm.impala"
        int _34788;
        _34788 = _34777 + bdimx_34641;
        #line 52 "impala/gpu_device_shm.impala"
        p_34777 = _34788;
        goto l34776;
    l34726: ;
        #line 243 "impala/gpu_device_shm.impala"
        int img_index_y_34743;
        img_index_y_34743 = _34694 + _34723;
        #line 249 "impala/gpu_device_shm.impala"
        int _34751;
        _34751 = img_index_y_34743 * _34689;
        #line 236 "impala/gpu_device_shm.impala"
        int shm_index_y_34735;
        shm_index_y_34735 = tidy_34656 + _34723;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34746;
        _34746 = img_index_y_34743 < _34697;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34744;
        _34744 = 0 <= img_index_y_34743;
        #line 248 "impala/gpu_device_shm.impala"
        int _34756;
        _34756 = shm_index_y_34735 * shm_dimx_34675;
        #line 239 "impala/gpu_device_shm.impala"
        bool _34736;
        _34736 = shm_index_y_34735 < shm_dimy_34666;
        #line 52 "impala/gpu_device_shm.impala"
        p_34728 = 0;
        goto l34727;
    l34727: ;
        _34728 = p_34728;
        #line 52 "impala/gpu_device_shm.impala"
        bool _34730;
        _34730 = _34728 < shm_dimx_34675;
        #line 52 "impala/gpu_device_shm.impala"
        if (_34730) goto l34731; else goto l34769;
    l34769: ;
        #line 56 "impala/gpu_device_shm.impala"
        int _34770;
        _34770 = _34723 + bdimy_34644;
        #line 52 "impala/gpu_device_shm.impala"
        p_34723 = _34770;
        goto l34722;
    l34731: ;
        #line 234 "impala/gpu_device_shm.impala"
        int shm_index_x_34732;
        shm_index_x_34732 = tidx_34653 + _34728;
        #line 239 "impala/gpu_device_shm.impala"
        bool _34733;
        _34733 = shm_index_x_34732 < shm_dimx_34675;
        #line 239 "impala/gpu_device_shm.impala"
        if (_34733) goto l34734; else goto l34768;
    l34768: ;
        #line 251 "impala/gpu_device_shm.impala"
        goto l34767;
    l34734: ;
        #line 239 "impala/gpu_device_shm.impala"
        if (_34736) goto l34737; else goto l34766;
    l34766: ;
        #line 251 "impala/gpu_device_shm.impala"
        goto l34767;
    l34767: ;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34748;
    l34737: ;
        #line 241 "impala/gpu_device_shm.impala"
        int img_index_x_34738;
        img_index_x_34738 = _34685 + _34728;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34739;
        _34739 = 0 <= img_index_x_34738;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34739) goto l34740; else goto l34765;
    l34765: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34762;
    l34740: ;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34741;
        _34741 = img_index_x_34738 < _34689;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34741) goto l34742; else goto l34764;
    l34764: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34762;
    l34742: ;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34744) goto l34745; else goto l34763;
    l34763: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34762;
    l34745: ;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34746) goto l34747; else goto l34761;
    l34761: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34762;
    l34762: ;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34748;
    l34747: ;
        #line 249 "impala/gpu_device_shm.impala"
        int _34752;
        _34752 = _34751 + img_index_x_34738;
        #line 249 "impala/gpu_device_shm.impala"
        double* _34753;
        _34753 = _28617_34638 + _34752;
        #line 248 "impala/gpu_device_shm.impala"
        int _34757;
        _34757 = _34756 + shm_index_x_34732;
        #line 249 "impala/gpu_device_shm.impala"
        double _34754;
        _34754 = *_34753;
        #line 248 "impala/gpu_device_shm.impala"
        double* _34758;
        _34758 = reserve_shared_34659 + _34757;
        #line 249 "impala/gpu_device_shm.impala"
        double _34759;
        _34759 = _34754;
        #line 248 "impala/gpu_device_shm.impala"
        *_34758 = _34759;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34748;
    l34748: ;
        #line 56 "impala/gpu_device_shm.impala"
        int _34750;
        _34750 = _34728 + bdimx_34641;
        #line 52 "impala/gpu_device_shm.impala"
        p_34728 = _34750;
        goto l34727;
    l34677: ;
        #line 234 "impala/gpu_device_shm.impala"
        int shm_index_x_34678;
        shm_index_x_34678 = tidx_34653 + _34670;
        #line 239 "impala/gpu_device_shm.impala"
        bool _34679;
        _34679 = shm_index_x_34678 < shm_dimx_34675;
        #line 239 "impala/gpu_device_shm.impala"
        if (_34679) goto l34680; else goto l34720;
    l34720: ;
        #line 251 "impala/gpu_device_shm.impala"
        goto l34719;
    l34680: ;
        #line 239 "impala/gpu_device_shm.impala"
        if (_34681) goto l34682; else goto l34718;
    l34718: ;
        #line 251 "impala/gpu_device_shm.impala"
        goto l34719;
    l34719: ;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34700;
    l34682: ;
        #line 241 "impala/gpu_device_shm.impala"
        int img_index_x_34686;
        img_index_x_34686 = _34685 + _34670;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34687;
        _34687 = 0 <= img_index_x_34686;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34687) goto l34688; else goto l34717;
    l34717: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34714;
    l34688: ;
        #line 246 "impala/gpu_device_shm.impala"
        bool _34690;
        _34690 = img_index_x_34686 < _34689;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34690) goto l34691; else goto l34716;
    l34716: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34714;
    l34691: ;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34695) goto l34696; else goto l34715;
    l34715: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34714;
    l34696: ;
        #line 246 "impala/gpu_device_shm.impala"
        if (_34698) goto l34699; else goto l34713;
    l34713: ;
        #line 250 "impala/gpu_device_shm.impala"
        goto l34714;
    l34714: ;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34700;
    l34699: ;
        #line 249 "impala/gpu_device_shm.impala"
        int _34704;
        _34704 = _34703 + img_index_x_34686;
        #line 248 "impala/gpu_device_shm.impala"
        int _34709;
        _34709 = _34708 + shm_index_x_34678;
        #line 249 "impala/gpu_device_shm.impala"
        double* _34705;
        _34705 = _28617_34638 + _34704;
        #line 248 "impala/gpu_device_shm.impala"
        double* _34710;
        _34710 = reserve_shared_34659 + _34709;
        #line 249 "impala/gpu_device_shm.impala"
        double _34706;
        _34706 = *_34705;
        #line 249 "impala/gpu_device_shm.impala"
        double _34711;
        _34711 = _34706;
        #line 248 "impala/gpu_device_shm.impala"
        *_34710 = _34711;
        #line 54 "impala/gpu_device_shm.impala"
        goto l34700;
    l34700: ;
        #line 56 "impala/gpu_device_shm.impala"
        int _34702;
        _34702 = _34670 + bdimx_34641;
        #line 52 "impala/gpu_device_shm.impala"
        p_34670 = _34702;
        goto l34669;
}

}
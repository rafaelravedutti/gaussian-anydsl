
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_6327;
typedef struct {
    struct_Buffer_6327 e0;
    struct_Buffer_6327 e1;
    int e2;
    int e3;
} struct_image_6326;
typedef struct {
    struct_Buffer_6327 e0;
    int e1;
    int e2;
} struct_filter_6332;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_25292(struct_image_6326, struct_Buffer_6327, double*, struct_filter_6332);
__global__ void lambda_25546(struct_image_6326, struct_filter_6332, double*, double*, struct_Buffer_6327);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_25292(struct_image_6326 _25295_30003, struct_Buffer_6327 _25296_30004, double* _25297_30005, struct_filter_6332 _25298_30006) {
    __shared__ double ds_img[134][7];
    int  bdimx_30012;
    int pbdimx_30012;
    int  bdimy_30018;
    int pbdimy_30018;
    int  bidx_30024;
    int pbidx_30024;
    int  bidy_30030;
    int pbidy_30030;
    int  tidx_30036;
    int ptidx_30036;
    int  tidy_30042;
    int ptidy_30042;
    double*  reserve_shared_30050;
    double* preserve_shared_30050;
    int  _30061;
    int p_30061;
    int  _30117;
    int p_30117;
    int  _30183;
    int p_30183;
    double  sum_30185;
    double psum_30185;
    int  _30122;
    int p_30122;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    bdimx_30012 = blockDim_x();
    pbdimx_30012 = bdimx_30012;
    l30010: ;
        bdimx_30012 = pbdimx_30012;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bdimy_30018 = blockDim_y();
        pbdimy_30018 = bdimy_30018;
    l30016: ;
        bdimy_30018 = pbdimy_30018;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bidx_30024 = blockIdx_x();
        pbidx_30024 = bidx_30024;
    l30022: ;
        bidx_30024 = pbidx_30024;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bidy_30030 = blockIdx_y();
        pbidy_30030 = bidy_30030;
    l30028: ;
        bidy_30030 = pbidy_30030;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        tidx_30036 = threadIdx_x();
        ptidx_30036 = tidx_30036;
    l30034: ;
        tidx_30036 = ptidx_30036;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        tidy_30042 = threadIdx_y();
        ptidy_30042 = tidy_30042;
    l30040: ;
        tidy_30042 = ptidy_30042;
        #line 201 "gpu_device_shm.impala"
        __shared__ double reserver_reserve_shared_30050[938];
        preserve_shared_30050 = reserver_reserve_shared_30050;
    l30048: ;
        reserve_shared_30050 = preserve_shared_30050;
        #line 193 "gpu_device_shm.impala"
        int _30054;
        _30054 = _25298_30006.e2;
        #line 189 "gpu_device_shm.impala"
        int _30084;
        _30084 = bidy_30030 * bdimy_30018;
        #line 11 "main.impala"
        int _30081;
        _30081 = _25295_30003.e2;
        #line 187 "gpu_device_shm.impala"
        int _30075;
        _30075 = bidx_30024 * bdimx_30012;
        #line 11 "main.impala"
        int _30090;
        _30090 = _25295_30003.e3;
        #line 4 "gaussian.impala"
        int _30064;
        _30064 = _25298_30006.e1;
        #line 193 "gpu_device_shm.impala"
        int extend_height_30055;
        extend_height_30055 = _30054 / 2;
        #line 189 "gpu_device_shm.impala"
        int gid_y_30085;
        gid_y_30085 = _30084 + tidy_30042;
        #line 187 "gpu_device_shm.impala"
        int gid_x_30076;
        gid_x_30076 = _30075 + tidx_30036;
        #line 4 "gaussian.impala"
        int h_anchor_30065;
        h_anchor_30065 = _30064 / 2;
        #line 197 "gpu_device_shm.impala"
        int _30056;
        _30056 = 2 * extend_height_30055;
        #line 195 "gpu_device_shm.impala"
        int _30066;
        _30066 = 2 * h_anchor_30065;
        #line 197 "gpu_device_shm.impala"
        int shm_dimy_30057;
        shm_dimy_30057 = bdimy_30018 + _30056;
        #line 195 "gpu_device_shm.impala"
        int shm_dimx_30067;
        shm_dimx_30067 = bdimx_30012 + _30066;
        #line 38 "gpu_device_shm.impala"
        bool _30058;
        _30058 = 0 < shm_dimy_30057;
        #line 38 "gpu_device_shm.impala"
        if (_30058) goto l30059; else goto l30234;
    l30234: ;
        #line 231 "gpu_device_shm.impala"
        goto l30166;
    l30059: ;
        #line 226 "gpu_device_shm.impala"
        int _30102;
        _30102 = tidy_30042 * shm_dimx_30067;
        #line 221 "gpu_device_shm.impala"
        int _30086;
        _30086 = gid_y_30085 - extend_height_30055;
        #line 219 "gpu_device_shm.impala"
        int _30077;
        _30077 = gid_x_30076 - h_anchor_30065;
        #line 217 "gpu_device_shm.impala"
        bool _30073;
        _30073 = tidy_30042 < shm_dimy_30057;
        #line 224 "gpu_device_shm.impala"
        bool _30087;
        _30087 = 0 <= _30086;
        #line 224 "gpu_device_shm.impala"
        bool _30091;
        _30091 = _30086 < _30090;
        #line 227 "gpu_device_shm.impala"
        int _30096;
        _30096 = _30086 * _30081;
        #line 38 "gpu_device_shm.impala"
        p_30061 = 0;
        goto l30060;
    l30060: ;
        _30061 = p_30061;
        #line 38 "gpu_device_shm.impala"
        bool _30068;
        _30068 = _30061 < shm_dimx_30067;
        #line 38 "gpu_device_shm.impala"
        if (_30068) goto l30069; else goto l30115;
    l30115: ;
        #line 38 "gpu_device_shm.impala"
        p_30117 = bdimy_30018;
        goto l30116;
    l30116: ;
        _30117 = p_30117;
        #line 38 "gpu_device_shm.impala"
        bool _30119;
        _30119 = _30117 < shm_dimy_30057;
        #line 38 "gpu_device_shm.impala"
        if (_30119) goto l30120; else goto l30165;
    l30165: ;
        #line 231 "gpu_device_shm.impala"
        goto l30166;
    l30166: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        __syncthreads();
    l30171: ;
        #line 237 "gpu_device_shm.impala"
        bool _30173;
        _30173 = gid_x_30076 < _30081;
        #line 237 "gpu_device_shm.impala"
        if (_30173) goto l30174; else goto l30233;
    l30233: ;
        #line 240 "gpu_device_shm.impala"
        goto l30232;
    l30174: ;
        #line 237 "gpu_device_shm.impala"
        bool _30175;
        _30175 = gid_y_30085 < _30090;
        #line 237 "gpu_device_shm.impala"
        if (_30175) goto l30176; else goto l30231;
    l30231: ;
        #line 240 "gpu_device_shm.impala"
        goto l30232;
    l30232: ;
        return ;
    l30176: ;
        #line 64 "gpu_device_shm.impala"
        char* _30213;
        _30213 = _25296_30004.e1;
        #line 204 "gpu_device_shm.impala"
        int _30198;
        _30198 = extend_height_30055 - _30084;
        #line 64 "gpu_device_shm.impala"
        double* _30214;
        union { double* dst; char* src; } u_30214;
        u_30214.src = _30213;
        _30214 = u_30214.dst;
        #line 203 "gpu_device_shm.impala"
        int _30203;
        _30203 = h_anchor_30065 - _30075;
        #line 64 "gpu_device_shm.impala"
        int _30215;
        _30215 = gid_y_30085 * _30081;
        #line 17 "gaussian.impala"
        bool _30177;
        _30177 = h_anchor_30065 <= gid_x_30076;
        #line 64 "gpu_device_shm.impala"
        int _30216;
        _30216 = _30215 + gid_x_30076;
        #line 72 "gpu_device_shm.impala"
        int _30199;
        _30199 = gid_y_30085 + _30198;
        #line 64 "gpu_device_shm.impala"
        double* _30217;
        _30217 = _30214 + _30216;
        #line 72 "gpu_device_shm.impala"
        int _30200;
        _30200 = _30199 * shm_dimx_30067;
        #line 17 "gaussian.impala"
        if (_30177) goto l30178; else goto l30230;
    l30230: ;
        #line 27 "gaussian.impala"
        goto l30222;
    l30178: ;
        #line 17 "gaussian.impala"
        int _30179;
        _30179 = _30081 - h_anchor_30065;
        #line 17 "gaussian.impala"
        bool _30180;
        _30180 = gid_x_30076 < _30179;
        #line 17 "gaussian.impala"
        if (_30180) goto l30181; else goto l30221;
    l30221: ;
        #line 27 "gaussian.impala"
        goto l30222;
    l30222: ;
        #line 72 "gpu_device_shm.impala"
        int _30223;
        _30223 = _30200 + gid_x_30076;
        #line 72 "gpu_device_shm.impala"
        int _30224;
        _30224 = _30223 + _30203;
        #line 72 "gpu_device_shm.impala"
        double* _30225;
        _30225 = reserve_shared_30050 + _30224;
        #line 72 "gpu_device_shm.impala"
        double _30226;
        _30226 = *_30225;
        #line 72 "gpu_device_shm.impala"
        double _30228;
        _30228 = _30226;
        #line 64 "gpu_device_shm.impala"
        *_30217 = _30228;
        return ;
    l30181: ;
        #line 77 "gpu_device_shm.impala"
        struct_Buffer_6327 _30191;
        _30191 = _25298_30006.e0;
        #line 19 "gaussian.impala"
        int _30219;
        _30219 = 0 - h_anchor_30065;
        #line 77 "gpu_device_shm.impala"
        char* _30192;
        _30192 = _30191.e1;
        #line 19 "gaussian.impala"
        int _30187;
        _30187 = 1 + h_anchor_30065;
        #line 77 "gpu_device_shm.impala"
        double* _30193;
        union { double* dst; char* src; } u_30193;
        u_30193.src = _30192;
        _30193 = u_30193.dst;
        #line 27 "gpu_device_shm.impala"
        p_30183 = _30219;
        psum_30185 = 0.000000e+00;
        goto l30182;
    l30182: ;
        _30183 = p_30183;
        sum_30185 = psum_30185;
        #line 27 "gpu_device_shm.impala"
        bool _30188;
        _30188 = _30183 < _30187;
        #line 27 "gpu_device_shm.impala"
        if (_30188) goto l30189; else goto l30212;
    l30212: ;
        #line 64 "gpu_device_shm.impala"
        *_30217 = sum_30185;
        return ;
    l30189: ;
        #line 31 "gpu_device_shm.impala"
        int _30190;
        _30190 = 1 + _30183;
        #line 21 "gaussian.impala"
        int _30201;
        _30201 = gid_x_30076 + _30183;
        #line 72 "gpu_device_shm.impala"
        int _30202;
        _30202 = _30200 + _30201;
        #line 21 "gaussian.impala"
        int _30194;
        _30194 = _30183 + h_anchor_30065;
        #line 72 "gpu_device_shm.impala"
        int _30204;
        _30204 = _30202 + _30203;
        #line 76 "gpu_device_shm.impala"
        double* i_30195;
        i_30195 = _30193 + _30194;
        #line 72 "gpu_device_shm.impala"
        double* _30205;
        _30205 = reserve_shared_30050 + _30204;
        #line 77 "gpu_device_shm.impala"
        double _30196;
        _30196 = *i_30195;
        #line 77 "gpu_device_shm.impala"
        double _30208;
        _30208 = _30196;
        #line 72 "gpu_device_shm.impala"
        double _30206;
        _30206 = *_30205;
        #line 72 "gpu_device_shm.impala"
        double _30209;
        _30209 = _30206;
        #line 21 "gaussian.impala"
        double _30210;
        _30210 = _30208 * _30209;
        #line 21 "gaussian.impala"
        double _30211;
        _30211 = sum_30185 + _30210;
        #line 27 "gpu_device_shm.impala"
        p_30183 = _30190;
        psum_30185 = _30211;
        goto l30182;
    l30120: ;
        #line 221 "gpu_device_shm.impala"
        int img_index_y_30137;
        img_index_y_30137 = _30086 + _30117;
        #line 214 "gpu_device_shm.impala"
        int shm_index_y_30129;
        shm_index_y_30129 = tidy_30042 + _30117;
        #line 224 "gpu_device_shm.impala"
        bool _30138;
        _30138 = 0 <= img_index_y_30137;
        #line 227 "gpu_device_shm.impala"
        int _30145;
        _30145 = img_index_y_30137 * _30081;
        #line 224 "gpu_device_shm.impala"
        bool _30140;
        _30140 = img_index_y_30137 < _30090;
        #line 217 "gpu_device_shm.impala"
        bool _30130;
        _30130 = shm_index_y_30129 < shm_dimy_30057;
        #line 226 "gpu_device_shm.impala"
        int _30150;
        _30150 = shm_index_y_30129 * shm_dimx_30067;
        #line 38 "gpu_device_shm.impala"
        p_30122 = 0;
        goto l30121;
    l30121: ;
        _30122 = p_30122;
        #line 38 "gpu_device_shm.impala"
        bool _30124;
        _30124 = _30122 < shm_dimx_30067;
        #line 38 "gpu_device_shm.impala"
        if (_30124) goto l30125; else goto l30163;
    l30163: ;
        #line 42 "gpu_device_shm.impala"
        int _30164;
        _30164 = _30117 + bdimy_30018;
        #line 38 "gpu_device_shm.impala"
        p_30117 = _30164;
        goto l30116;
    l30125: ;
        #line 212 "gpu_device_shm.impala"
        int shm_index_x_30126;
        shm_index_x_30126 = tidx_30036 + _30122;
        #line 217 "gpu_device_shm.impala"
        bool _30127;
        _30127 = shm_index_x_30126 < shm_dimx_30067;
        #line 217 "gpu_device_shm.impala"
        if (_30127) goto l30128; else goto l30162;
    l30162: ;
        #line 229 "gpu_device_shm.impala"
        goto l30161;
    l30128: ;
        #line 217 "gpu_device_shm.impala"
        if (_30130) goto l30131; else goto l30160;
    l30160: ;
        #line 229 "gpu_device_shm.impala"
        goto l30161;
    l30161: ;
        #line 40 "gpu_device_shm.impala"
        goto l30142;
    l30131: ;
        #line 219 "gpu_device_shm.impala"
        int img_index_x_30132;
        img_index_x_30132 = _30077 + _30122;
        #line 224 "gpu_device_shm.impala"
        bool _30133;
        _30133 = 0 <= img_index_x_30132;
        #line 224 "gpu_device_shm.impala"
        if (_30133) goto l30134; else goto l30159;
    l30159: ;
        #line 228 "gpu_device_shm.impala"
        goto l30156;
    l30134: ;
        #line 224 "gpu_device_shm.impala"
        bool _30135;
        _30135 = img_index_x_30132 < _30081;
        #line 224 "gpu_device_shm.impala"
        if (_30135) goto l30136; else goto l30158;
    l30158: ;
        #line 228 "gpu_device_shm.impala"
        goto l30156;
    l30136: ;
        #line 224 "gpu_device_shm.impala"
        if (_30138) goto l30139; else goto l30157;
    l30157: ;
        #line 228 "gpu_device_shm.impala"
        goto l30156;
    l30139: ;
        #line 224 "gpu_device_shm.impala"
        if (_30140) goto l30141; else goto l30155;
    l30155: ;
        #line 228 "gpu_device_shm.impala"
        goto l30156;
    l30156: ;
        #line 40 "gpu_device_shm.impala"
        goto l30142;
    l30141: ;
        #line 226 "gpu_device_shm.impala"
        int _30151;
        _30151 = _30150 + shm_index_x_30126;
        #line 226 "gpu_device_shm.impala"
        double* _30152;
        _30152 = reserve_shared_30050 + _30151;
        #line 227 "gpu_device_shm.impala"
        int _30146;
        _30146 = _30145 + img_index_x_30132;
        #line 227 "gpu_device_shm.impala"
        double* _30147;
        _30147 = _25297_30005 + _30146;
        #line 227 "gpu_device_shm.impala"
        double _30148;
        _30148 = *_30147;
        #line 227 "gpu_device_shm.impala"
        double _30153;
        _30153 = _30148;
        #line 226 "gpu_device_shm.impala"
        *_30152 = _30153;
        #line 40 "gpu_device_shm.impala"
        goto l30142;
    l30142: ;
        #line 42 "gpu_device_shm.impala"
        int _30144;
        _30144 = _30122 + bdimx_30012;
        #line 38 "gpu_device_shm.impala"
        p_30122 = _30144;
        goto l30121;
    l30069: ;
        #line 212 "gpu_device_shm.impala"
        int shm_index_x_30070;
        shm_index_x_30070 = tidx_30036 + _30061;
        #line 217 "gpu_device_shm.impala"
        bool _30071;
        _30071 = shm_index_x_30070 < shm_dimx_30067;
        #line 217 "gpu_device_shm.impala"
        if (_30071) goto l30072; else goto l30114;
    l30114: ;
        #line 229 "gpu_device_shm.impala"
        goto l30113;
    l30072: ;
        #line 217 "gpu_device_shm.impala"
        if (_30073) goto l30074; else goto l30112;
    l30112: ;
        #line 229 "gpu_device_shm.impala"
        goto l30113;
    l30113: ;
        #line 40 "gpu_device_shm.impala"
        goto l30093;
    l30074: ;
        #line 219 "gpu_device_shm.impala"
        int img_index_x_30078;
        img_index_x_30078 = _30077 + _30061;
        #line 224 "gpu_device_shm.impala"
        bool _30079;
        _30079 = 0 <= img_index_x_30078;
        #line 224 "gpu_device_shm.impala"
        if (_30079) goto l30080; else goto l30111;
    l30111: ;
        #line 228 "gpu_device_shm.impala"
        goto l30108;
    l30080: ;
        #line 224 "gpu_device_shm.impala"
        bool _30082;
        _30082 = img_index_x_30078 < _30081;
        #line 224 "gpu_device_shm.impala"
        if (_30082) goto l30083; else goto l30110;
    l30110: ;
        #line 228 "gpu_device_shm.impala"
        goto l30108;
    l30083: ;
        #line 224 "gpu_device_shm.impala"
        if (_30087) goto l30088; else goto l30109;
    l30109: ;
        #line 228 "gpu_device_shm.impala"
        goto l30108;
    l30088: ;
        #line 224 "gpu_device_shm.impala"
        if (_30091) goto l30092; else goto l30107;
    l30107: ;
        #line 228 "gpu_device_shm.impala"
        goto l30108;
    l30108: ;
        #line 40 "gpu_device_shm.impala"
        goto l30093;
    l30092: ;
        #line 226 "gpu_device_shm.impala"
        int _30103;
        _30103 = _30102 + shm_index_x_30070;
        #line 226 "gpu_device_shm.impala"
        double* _30104;
        _30104 = reserve_shared_30050 + _30103;
        #line 227 "gpu_device_shm.impala"
        int _30097;
        _30097 = _30096 + img_index_x_30078;
        #line 227 "gpu_device_shm.impala"
        double* _30098;
        _30098 = _25297_30005 + _30097;
        #line 227 "gpu_device_shm.impala"
        double _30099;
        _30099 = *_30098;
        #line 227 "gpu_device_shm.impala"
        double _30105;
        _30105 = _30099;
        #line 226 "gpu_device_shm.impala"
        *_30104 = _30105;
        #line 40 "gpu_device_shm.impala"
        goto l30093;
    l30093: ;
        #line 42 "gpu_device_shm.impala"
        int _30095;
        _30095 = _30061 + bdimx_30012;
        #line 38 "gpu_device_shm.impala"
        p_30061 = _30095;
        goto l30060;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_25546(struct_image_6326 _25549_30238, struct_filter_6332 _25550_30239, double* _25551_30240, double* _25552_30241, struct_Buffer_6327 _25553_30242) {
    __shared__ double ds_img[134][7];
    int  bdimx_30245;
    int pbdimx_30245;
    int  bdimy_30248;
    int pbdimy_30248;
    int  bidx_30251;
    int pbidx_30251;
    int  bidy_30254;
    int pbidy_30254;
    int  tidx_30257;
    int ptidx_30257;
    int  tidy_30260;
    int ptidy_30260;
    double*  reserve_shared_30263;
    double* preserve_shared_30263;
    int  _30271;
    int p_30271;
    int  _30324;
    int p_30324;
    int  _30387;
    int p_30387;
    double  sum_30389;
    double psum_30389;
    int  _30329;
    int p_30329;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    bdimx_30245 = blockDim_x();
    pbdimx_30245 = bdimx_30245;
    l30243: ;
        bdimx_30245 = pbdimx_30245;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bdimy_30248 = blockDim_y();
        pbdimy_30248 = bdimy_30248;
    l30246: ;
        bdimy_30248 = pbdimy_30248;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bidx_30251 = blockIdx_x();
        pbidx_30251 = bidx_30251;
    l30249: ;
        bidx_30251 = pbidx_30251;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bidy_30254 = blockIdx_y();
        pbidy_30254 = bidy_30254;
    l30252: ;
        bidy_30254 = pbidy_30254;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        tidx_30257 = threadIdx_x();
        ptidx_30257 = tidx_30257;
    l30255: ;
        tidx_30257 = ptidx_30257;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        tidy_30260 = threadIdx_y();
        ptidy_30260 = tidy_30260;
    l30258: ;
        tidy_30260 = ptidy_30260;
        #line 201 "gpu_device_shm.impala"
        __shared__ double reserver_reserve_shared_30263[938];
        preserve_shared_30263 = reserver_reserve_shared_30263;
    l30261: ;
        reserve_shared_30263 = preserve_shared_30263;
        #line 187 "gpu_device_shm.impala"
        int _30284;
        _30284 = bidx_30251 * bdimx_30245;
        #line 11 "main.impala"
        int _30298;
        _30298 = _25549_30238.e3;
        #line 187 "gpu_device_shm.impala"
        int gid_x_30285;
        gid_x_30285 = _30284 + tidx_30257;
        #line 6 "gaussian.impala"
        int _30264;
        _30264 = _25550_30239.e2;
        #line 191 "gpu_device_shm.impala"
        int _30273;
        _30273 = _25550_30239.e1;
        #line 189 "gpu_device_shm.impala"
        int _30293;
        _30293 = bidy_30254 * bdimy_30248;
        #line 11 "main.impala"
        int _30290;
        _30290 = _25549_30238.e2;
        #line 189 "gpu_device_shm.impala"
        int gid_y_30294;
        gid_y_30294 = _30293 + tidy_30260;
        #line 191 "gpu_device_shm.impala"
        int extend_width_30274;
        extend_width_30274 = _30273 / 2;
        #line 6 "gaussian.impala"
        int v_anchor_30265;
        v_anchor_30265 = _30264 / 2;
        #line 195 "gpu_device_shm.impala"
        int _30275;
        _30275 = 2 * extend_width_30274;
        #line 197 "gpu_device_shm.impala"
        int _30266;
        _30266 = 2 * v_anchor_30265;
        #line 195 "gpu_device_shm.impala"
        int shm_dimx_30276;
        shm_dimx_30276 = bdimx_30245 + _30275;
        #line 197 "gpu_device_shm.impala"
        int shm_dimy_30267;
        shm_dimy_30267 = bdimy_30248 + _30266;
        #line 38 "gpu_device_shm.impala"
        bool _30268;
        _30268 = 0 < shm_dimy_30267;
        #line 38 "gpu_device_shm.impala"
        if (_30268) goto l30269; else goto l30438;
    l30438: ;
        #line 231 "gpu_device_shm.impala"
        goto l30373;
    l30269: ;
        #line 226 "gpu_device_shm.impala"
        int _30309;
        _30309 = tidy_30260 * shm_dimx_30276;
        #line 219 "gpu_device_shm.impala"
        int _30286;
        _30286 = gid_x_30285 - extend_width_30274;
        #line 221 "gpu_device_shm.impala"
        int _30295;
        _30295 = gid_y_30294 - v_anchor_30265;
        #line 217 "gpu_device_shm.impala"
        bool _30282;
        _30282 = tidy_30260 < shm_dimy_30267;
        #line 224 "gpu_device_shm.impala"
        bool _30296;
        _30296 = 0 <= _30295;
        #line 224 "gpu_device_shm.impala"
        bool _30299;
        _30299 = _30295 < _30298;
        #line 227 "gpu_device_shm.impala"
        int _30304;
        _30304 = _30295 * _30290;
        #line 38 "gpu_device_shm.impala"
        p_30271 = 0;
        goto l30270;
    l30270: ;
        _30271 = p_30271;
        #line 38 "gpu_device_shm.impala"
        bool _30277;
        _30277 = _30271 < shm_dimx_30276;
        #line 38 "gpu_device_shm.impala"
        if (_30277) goto l30278; else goto l30322;
    l30322: ;
        #line 38 "gpu_device_shm.impala"
        p_30324 = bdimy_30248;
        goto l30323;
    l30323: ;
        _30324 = p_30324;
        #line 38 "gpu_device_shm.impala"
        bool _30326;
        _30326 = _30324 < shm_dimy_30267;
        #line 38 "gpu_device_shm.impala"
        if (_30326) goto l30327; else goto l30372;
    l30372: ;
        #line 231 "gpu_device_shm.impala"
        goto l30373;
    l30373: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        __syncthreads();
    l30375: ;
        #line 237 "gpu_device_shm.impala"
        bool _30377;
        _30377 = gid_x_30285 < _30290;
        #line 237 "gpu_device_shm.impala"
        if (_30377) goto l30378; else goto l30437;
    l30437: ;
        #line 240 "gpu_device_shm.impala"
        goto l30436;
    l30378: ;
        #line 237 "gpu_device_shm.impala"
        bool _30379;
        _30379 = gid_y_30294 < _30298;
        #line 237 "gpu_device_shm.impala"
        if (_30379) goto l30380; else goto l30435;
    l30435: ;
        #line 240 "gpu_device_shm.impala"
        goto l30436;
    l30436: ;
        return ;
    l30380: ;
        #line 204 "gpu_device_shm.impala"
        int _30402;
        _30402 = v_anchor_30265 - _30293;
        #line 64 "gpu_device_shm.impala"
        char* _30416;
        _30416 = _25553_30242.e1;
        #line 203 "gpu_device_shm.impala"
        int _30406;
        _30406 = extend_width_30274 - _30284;
        #line 64 "gpu_device_shm.impala"
        int _30418;
        _30418 = gid_y_30294 * _30290;
        #line 64 "gpu_device_shm.impala"
        int _30419;
        _30419 = _30418 + gid_x_30285;
        #line 64 "gpu_device_shm.impala"
        double* _30417;
        union { double* dst; char* src; } u_30417;
        u_30417.src = _30416;
        _30417 = u_30417.dst;
        #line 39 "gaussian.impala"
        bool _30381;
        _30381 = v_anchor_30265 <= gid_y_30294;
        #line 64 "gpu_device_shm.impala"
        double* _30420;
        _30420 = _30417 + _30419;
        #line 39 "gaussian.impala"
        if (_30381) goto l30382; else goto l30434;
    l30434: ;
        #line 49 "gaussian.impala"
        goto l30424;
    l30382: ;
        #line 39 "gaussian.impala"
        int _30383;
        _30383 = _30298 - v_anchor_30265;
        #line 39 "gaussian.impala"
        bool _30384;
        _30384 = gid_y_30294 < _30383;
        #line 39 "gaussian.impala"
        if (_30384) goto l30385; else goto l30423;
    l30423: ;
        #line 49 "gaussian.impala"
        goto l30424;
    l30424: ;
        #line 72 "gpu_device_shm.impala"
        int _30425;
        _30425 = gid_y_30294 + _30402;
        #line 72 "gpu_device_shm.impala"
        int _30426;
        _30426 = _30425 * shm_dimx_30276;
        #line 72 "gpu_device_shm.impala"
        int _30427;
        _30427 = _30426 + gid_x_30285;
        #line 72 "gpu_device_shm.impala"
        int _30428;
        _30428 = _30427 + _30406;
        #line 72 "gpu_device_shm.impala"
        double* _30429;
        _30429 = reserve_shared_30263 + _30428;
        #line 72 "gpu_device_shm.impala"
        double _30430;
        _30430 = *_30429;
        #line 72 "gpu_device_shm.impala"
        double _30432;
        _30432 = _30430;
        #line 64 "gpu_device_shm.impala"
        *_30420 = _30432;
        return ;
    l30385: ;
        #line 41 "gaussian.impala"
        int _30422;
        _30422 = 0 - v_anchor_30265;
        #line 77 "gpu_device_shm.impala"
        struct_Buffer_6327 _30394;
        _30394 = _25550_30239.e0;
        #line 41 "gaussian.impala"
        int _30390;
        _30390 = 1 + v_anchor_30265;
        #line 77 "gpu_device_shm.impala"
        char* _30395;
        _30395 = _30394.e1;
        #line 77 "gpu_device_shm.impala"
        double* _30396;
        union { double* dst; char* src; } u_30396;
        u_30396.src = _30395;
        _30396 = u_30396.dst;
        #line 27 "gpu_device_shm.impala"
        p_30387 = _30422;
        psum_30389 = 0.000000e+00;
        goto l30386;
    l30386: ;
        _30387 = p_30387;
        sum_30389 = psum_30389;
        #line 27 "gpu_device_shm.impala"
        bool _30391;
        _30391 = _30387 < _30390;
        #line 27 "gpu_device_shm.impala"
        if (_30391) goto l30392; else goto l30415;
    l30415: ;
        #line 64 "gpu_device_shm.impala"
        *_30420 = sum_30389;
        return ;
    l30392: ;
        #line 43 "gaussian.impala"
        int _30401;
        _30401 = gid_y_30294 + _30387;
        #line 43 "gaussian.impala"
        int _30397;
        _30397 = _30387 + v_anchor_30265;
        #line 31 "gpu_device_shm.impala"
        int _30393;
        _30393 = 1 + _30387;
        #line 72 "gpu_device_shm.impala"
        int _30403;
        _30403 = _30401 + _30402;
        #line 76 "gpu_device_shm.impala"
        double* i_30398;
        i_30398 = _30396 + _30397;
        #line 72 "gpu_device_shm.impala"
        int _30404;
        _30404 = _30403 * shm_dimx_30276;
        #line 77 "gpu_device_shm.impala"
        double _30399;
        _30399 = *i_30398;
        #line 72 "gpu_device_shm.impala"
        int _30405;
        _30405 = _30404 + gid_x_30285;
        #line 77 "gpu_device_shm.impala"
        double _30411;
        _30411 = _30399;
        #line 72 "gpu_device_shm.impala"
        int _30407;
        _30407 = _30405 + _30406;
        #line 72 "gpu_device_shm.impala"
        double* _30408;
        _30408 = reserve_shared_30263 + _30407;
        #line 72 "gpu_device_shm.impala"
        double _30409;
        _30409 = *_30408;
        #line 72 "gpu_device_shm.impala"
        double _30412;
        _30412 = _30409;
        #line 43 "gaussian.impala"
        double _30413;
        _30413 = _30411 * _30412;
        #line 43 "gaussian.impala"
        double _30414;
        _30414 = sum_30389 + _30413;
        #line 27 "gpu_device_shm.impala"
        p_30387 = _30393;
        psum_30389 = _30414;
        goto l30386;
    l30327: ;
        #line 221 "gpu_device_shm.impala"
        int img_index_y_30344;
        img_index_y_30344 = _30295 + _30324;
        #line 224 "gpu_device_shm.impala"
        bool _30347;
        _30347 = img_index_y_30344 < _30298;
        #line 224 "gpu_device_shm.impala"
        bool _30345;
        _30345 = 0 <= img_index_y_30344;
        #line 214 "gpu_device_shm.impala"
        int shm_index_y_30336;
        shm_index_y_30336 = tidy_30260 + _30324;
        #line 227 "gpu_device_shm.impala"
        int _30352;
        _30352 = img_index_y_30344 * _30290;
        #line 217 "gpu_device_shm.impala"
        bool _30337;
        _30337 = shm_index_y_30336 < shm_dimy_30267;
        #line 226 "gpu_device_shm.impala"
        int _30357;
        _30357 = shm_index_y_30336 * shm_dimx_30276;
        #line 38 "gpu_device_shm.impala"
        p_30329 = 0;
        goto l30328;
    l30328: ;
        _30329 = p_30329;
        #line 38 "gpu_device_shm.impala"
        bool _30331;
        _30331 = _30329 < shm_dimx_30276;
        #line 38 "gpu_device_shm.impala"
        if (_30331) goto l30332; else goto l30370;
    l30370: ;
        #line 42 "gpu_device_shm.impala"
        int _30371;
        _30371 = _30324 + bdimy_30248;
        #line 38 "gpu_device_shm.impala"
        p_30324 = _30371;
        goto l30323;
    l30332: ;
        #line 212 "gpu_device_shm.impala"
        int shm_index_x_30333;
        shm_index_x_30333 = tidx_30257 + _30329;
        #line 217 "gpu_device_shm.impala"
        bool _30334;
        _30334 = shm_index_x_30333 < shm_dimx_30276;
        #line 217 "gpu_device_shm.impala"
        if (_30334) goto l30335; else goto l30369;
    l30369: ;
        #line 229 "gpu_device_shm.impala"
        goto l30368;
    l30335: ;
        #line 217 "gpu_device_shm.impala"
        if (_30337) goto l30338; else goto l30367;
    l30367: ;
        #line 229 "gpu_device_shm.impala"
        goto l30368;
    l30368: ;
        #line 40 "gpu_device_shm.impala"
        goto l30349;
    l30338: ;
        #line 219 "gpu_device_shm.impala"
        int img_index_x_30339;
        img_index_x_30339 = _30286 + _30329;
        #line 224 "gpu_device_shm.impala"
        bool _30340;
        _30340 = 0 <= img_index_x_30339;
        #line 224 "gpu_device_shm.impala"
        if (_30340) goto l30341; else goto l30366;
    l30366: ;
        #line 228 "gpu_device_shm.impala"
        goto l30363;
    l30341: ;
        #line 224 "gpu_device_shm.impala"
        bool _30342;
        _30342 = img_index_x_30339 < _30290;
        #line 224 "gpu_device_shm.impala"
        if (_30342) goto l30343; else goto l30365;
    l30365: ;
        #line 228 "gpu_device_shm.impala"
        goto l30363;
    l30343: ;
        #line 224 "gpu_device_shm.impala"
        if (_30345) goto l30346; else goto l30364;
    l30364: ;
        #line 228 "gpu_device_shm.impala"
        goto l30363;
    l30346: ;
        #line 224 "gpu_device_shm.impala"
        if (_30347) goto l30348; else goto l30362;
    l30362: ;
        #line 228 "gpu_device_shm.impala"
        goto l30363;
    l30363: ;
        #line 40 "gpu_device_shm.impala"
        goto l30349;
    l30348: ;
        #line 227 "gpu_device_shm.impala"
        int _30353;
        _30353 = _30352 + img_index_x_30339;
        #line 227 "gpu_device_shm.impala"
        double* _30354;
        _30354 = _25551_30240 + _30353;
        #line 227 "gpu_device_shm.impala"
        double _30355;
        _30355 = *_30354;
        #line 226 "gpu_device_shm.impala"
        int _30358;
        _30358 = _30357 + shm_index_x_30333;
        #line 227 "gpu_device_shm.impala"
        double _30360;
        _30360 = _30355;
        #line 226 "gpu_device_shm.impala"
        double* _30359;
        _30359 = reserve_shared_30263 + _30358;
        #line 226 "gpu_device_shm.impala"
        *_30359 = _30360;
        #line 40 "gpu_device_shm.impala"
        goto l30349;
    l30349: ;
        #line 42 "gpu_device_shm.impala"
        int _30351;
        _30351 = _30329 + bdimx_30245;
        #line 38 "gpu_device_shm.impala"
        p_30329 = _30351;
        goto l30328;
    l30278: ;
        #line 212 "gpu_device_shm.impala"
        int shm_index_x_30279;
        shm_index_x_30279 = tidx_30257 + _30271;
        #line 217 "gpu_device_shm.impala"
        bool _30280;
        _30280 = shm_index_x_30279 < shm_dimx_30276;
        #line 217 "gpu_device_shm.impala"
        if (_30280) goto l30281; else goto l30321;
    l30321: ;
        #line 229 "gpu_device_shm.impala"
        goto l30320;
    l30281: ;
        #line 217 "gpu_device_shm.impala"
        if (_30282) goto l30283; else goto l30319;
    l30319: ;
        #line 229 "gpu_device_shm.impala"
        goto l30320;
    l30320: ;
        #line 40 "gpu_device_shm.impala"
        goto l30301;
    l30283: ;
        #line 219 "gpu_device_shm.impala"
        int img_index_x_30287;
        img_index_x_30287 = _30286 + _30271;
        #line 224 "gpu_device_shm.impala"
        bool _30288;
        _30288 = 0 <= img_index_x_30287;
        #line 224 "gpu_device_shm.impala"
        if (_30288) goto l30289; else goto l30318;
    l30318: ;
        #line 228 "gpu_device_shm.impala"
        goto l30315;
    l30289: ;
        #line 224 "gpu_device_shm.impala"
        bool _30291;
        _30291 = img_index_x_30287 < _30290;
        #line 224 "gpu_device_shm.impala"
        if (_30291) goto l30292; else goto l30317;
    l30317: ;
        #line 228 "gpu_device_shm.impala"
        goto l30315;
    l30292: ;
        #line 224 "gpu_device_shm.impala"
        if (_30296) goto l30297; else goto l30316;
    l30316: ;
        #line 228 "gpu_device_shm.impala"
        goto l30315;
    l30297: ;
        #line 224 "gpu_device_shm.impala"
        if (_30299) goto l30300; else goto l30314;
    l30314: ;
        #line 228 "gpu_device_shm.impala"
        goto l30315;
    l30315: ;
        #line 40 "gpu_device_shm.impala"
        goto l30301;
    l30300: ;
        #line 226 "gpu_device_shm.impala"
        int _30310;
        _30310 = _30309 + shm_index_x_30279;
        #line 227 "gpu_device_shm.impala"
        int _30305;
        _30305 = _30304 + img_index_x_30287;
        #line 226 "gpu_device_shm.impala"
        double* _30311;
        _30311 = reserve_shared_30263 + _30310;
        #line 227 "gpu_device_shm.impala"
        double* _30306;
        _30306 = _25551_30240 + _30305;
        #line 227 "gpu_device_shm.impala"
        double _30307;
        _30307 = *_30306;
        #line 227 "gpu_device_shm.impala"
        double _30312;
        _30312 = _30307;
        #line 226 "gpu_device_shm.impala"
        *_30311 = _30312;
        #line 40 "gpu_device_shm.impala"
        goto l30301;
    l30301: ;
        #line 42 "gpu_device_shm.impala"
        int _30303;
        _30303 = _30271 + bdimx_30245;
        #line 38 "gpu_device_shm.impala"
        p_30271 = _30303;
        goto l30270;
}

}

#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_4844;
typedef struct {
    struct_Buffer_4844 e0;
    struct_Buffer_4844 e1;
    int e2;
    int e3;
} struct_image_4847;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_17870(double*, struct_Buffer_4844, struct_image_4847, double*);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_17870(double* _17873_19351, struct_Buffer_4844 _17874_19352, struct_image_4847 _17875_19353, double* _17876_19354) {
    int  _19360;
    int p_19360;
    int  _19366;
    int p_19366;
    int  _19372;
    int p_19372;
    int  _19378;
    int p_19378;
    int  _19384;
    int p_19384;
    int  _19390;
    int p_19390;
    int  _19415;
    int p_19415;
    double  sum_19417;
    double psum_19417;
    int  _19422;
    int p_19422;
    double  sum_19424;
    double psum_19424;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _19360 = blockIdx_x();
    p_19360 = _19360;
    l19358: ;
        _19360 = p_19360;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19366 = blockDim_x();
        p_19366 = _19366;
    l19364: ;
        _19366 = p_19366;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19372 = threadIdx_x();
        p_19372 = _19372;
    l19370: ;
        _19372 = p_19372;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19378 = blockIdx_y();
        p_19378 = _19378;
    l19376: ;
        _19378 = p_19378;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19384 = blockDim_y();
        p_19384 = _19384;
    l19382: ;
        _19384 = p_19384;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19390 = threadIdx_y();
        p_19390 = _19390;
    l19388: ;
        _19390 = p_19390;
        #line 11 "main.impala"
        int _19394;
        _19394 = _17875_19353.e2;
        #line 119 "gpu_device.impala"
        int _19391;
        _19391 = _19360 * _19366;
        #line 119 "gpu_device.impala"
        int gid_x_19392;
        gid_x_19392 = _19391 + _19372;
        #line 124 "gpu_device.impala"
        bool _19395;
        _19395 = gid_x_19392 < _19394;
        #line 124 "gpu_device.impala"
        if (_19395) goto l19396; else goto l19476;
    l19476: ;
        #line 127 "gpu_device.impala"
        goto l19475;
    l19396: ;
        #line 121 "gpu_device.impala"
        int _19397;
        _19397 = _19378 * _19384;
        #line 121 "gpu_device.impala"
        int gid_y_19398;
        gid_y_19398 = _19397 + _19390;
        #line 11 "main.impala"
        int _19400;
        _19400 = _17875_19353.e3;
        #line 124 "gpu_device.impala"
        bool _19401;
        _19401 = gid_y_19398 < _19400;
        #line 124 "gpu_device.impala"
        if (_19401) goto l19402; else goto l19474;
    l19474: ;
        #line 127 "gpu_device.impala"
        goto l19475;
    l19475: ;
        return ;
    l19402: ;
        #line 68 "gaussian.impala"
        bool _19404;
        _19404 = 3 <= gid_x_19392;
        #line 49 "gpu_device.impala"
        int _19459;
        _19459 = gid_y_19398 * _19394;
        #line 49 "gpu_device.impala"
        int _19460;
        _19460 = _19459 + gid_x_19392;
        #line 44 "gpu_device.impala"
        char* _19457;
        _19457 = _17874_19352.e1;
        #line 49 "gpu_device.impala"
        struct_Buffer_4844 _19439;
        _19439 = _17875_19353.e1;
        #line 44 "gpu_device.impala"
        double* _19458;
        union { double* dst; char* src; } u_19458;
        u_19458.src = _19457;
        _19458 = u_19458.dst;
        #line 44 "gpu_device.impala"
        double* _19461;
        _19461 = _19458 + _19460;
        #line 49 "gpu_device.impala"
        char* _19440;
        _19440 = _19439.e1;
        #line 49 "gpu_device.impala"
        double* _19441;
        union { double* dst; char* src; } u_19441;
        u_19441.src = _19440;
        _19441 = u_19441.dst;
        #line 68 "gaussian.impala"
        if (_19404) goto l19405; else goto l19473;
    l19473: ;
        #line 80 "gaussian.impala"
        goto l19465;
    l19405: ;
        #line 68 "gaussian.impala"
        int _19406;
        _19406 = _19394 - 3;
        #line 68 "gaussian.impala"
        bool _19407;
        _19407 = gid_x_19392 < _19406;
        #line 68 "gaussian.impala"
        if (_19407) goto l19408; else goto l19472;
    l19472: ;
        #line 80 "gaussian.impala"
        goto l19465;
    l19408: ;
        #line 68 "gaussian.impala"
        bool _19409;
        _19409 = 3 <= gid_y_19398;
        #line 68 "gaussian.impala"
        if (_19409) goto l19410; else goto l19471;
    l19471: ;
        #line 80 "gaussian.impala"
        goto l19465;
    l19410: ;
        #line 68 "gaussian.impala"
        int _19411;
        _19411 = _19400 - 3;
        #line 68 "gaussian.impala"
        bool _19412;
        _19412 = gid_y_19398 < _19411;
        #line 68 "gaussian.impala"
        if (_19412) goto l19413; else goto l19464;
    l19464: ;
        #line 80 "gaussian.impala"
        goto l19465;
    l19465: ;
        #line 49 "gpu_device.impala"
        double* _19466;
        _19466 = _19441 + _19460;
        #line 49 "gpu_device.impala"
        double _19467;
        _19467 = *_19466;
        #line 49 "gpu_device.impala"
        double _19469;
        _19469 = _19467;
        #line 44 "gpu_device.impala"
        *_19461 = _19469;
        return ;
    l19413: ;
        #line 18 "gpu_device.impala"
        p_19415 = -3;
        psum_19417 = 0.000000e+00;
        goto l19414;
    l19414: ;
        _19415 = p_19415;
        sum_19417 = psum_19417;
        #line 18 "gpu_device.impala"
        bool _19419;
        _19419 = _19415 < 4;
        #line 18 "gpu_device.impala"
        if (_19419) goto l19420; else goto l19456;
    l19456: ;
        #line 44 "gpu_device.impala"
        *_19461 = sum_19417;
        return ;
    l19420: ;
        #line 73 "gaussian.impala"
        int _19444;
        _19444 = gid_x_19392 + _19415;
        #line 18 "gpu_device.impala"
        p_19422 = -3;
        psum_19424 = sum_19417;
        goto l19421;
    l19421: ;
        _19422 = p_19422;
        sum_19424 = psum_19424;
        #line 18 "gpu_device.impala"
        bool _19425;
        _19425 = _19422 < 4;
        #line 18 "gpu_device.impala"
        if (_19425) goto l19426; else goto l19453;
    l19453: ;
        #line 22 "gpu_device.impala"
        int _19454;
        _19454 = 1 + _19415;
        #line 18 "gpu_device.impala"
        p_19415 = _19454;
        psum_19417 = sum_19424;
        goto l19414;
    l19426: ;
        #line 73 "gaussian.impala"
        int _19442;
        _19442 = gid_y_19398 + _19422;
        #line 73 "gaussian.impala"
        int _19430;
        _19430 = 3 + _19422;
        #line 22 "gpu_device.impala"
        int _19428;
        _19428 = 1 + _19422;
        #line 59 "gpu_device.impala"
        int _19431;
        _19431 = 7 * _19430;
        #line 49 "gpu_device.impala"
        int _19443;
        _19443 = _19442 * _19394;
        #line 59 "gpu_device.impala"
        int _19432;
        _19432 = _19431 + _19415;
        #line 49 "gpu_device.impala"
        int _19445;
        _19445 = _19443 + _19444;
        #line 59 "gpu_device.impala"
        int _19433;
        _19433 = 3 + _19432;
        #line 49 "gpu_device.impala"
        double* _19446;
        _19446 = _19441 + _19445;
        #line 59 "gpu_device.impala"
        double* _19434;
        _19434 = _17876_19354 + _19433;
        #line 59 "gpu_device.impala"
        double _19435;
        _19435 = *_19434;
        #line 59 "gpu_device.impala"
        double _19449;
        _19449 = _19435;
        #line 49 "gpu_device.impala"
        double _19447;
        _19447 = *_19446;
        #line 49 "gpu_device.impala"
        double _19450;
        _19450 = _19447;
        #line 73 "gaussian.impala"
        double _19451;
        _19451 = _19449 * _19450;
        #line 73 "gaussian.impala"
        double _19452;
        _19452 = sum_19424 + _19451;
        #line 18 "gpu_device.impala"
        p_19422 = _19428;
        psum_19424 = _19452;
        goto l19421;
}

}
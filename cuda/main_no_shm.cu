
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_5425;
typedef struct {
    struct_Buffer_5425 e0;
    struct_Buffer_5425 e1;
    int e2;
    int e3;
} struct_image_5424;
typedef struct {
    struct_Buffer_5425 e0;
    int e1;
    int e2;
} struct_filter_5428;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_20643(struct_image_5424, struct_filter_5428, struct_Buffer_5425);
__global__ void lambda_20775(struct_filter_5428, struct_image_5424, struct_Buffer_5425, double*, struct_Buffer_5425);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_20643(struct_image_5424 _20646_22982, struct_filter_5428 _20647_22983, struct_Buffer_5425 _20648_22984) {
    __shared__ double ds_img[134][7];
    int  _22990;
    int p_22990;
    int  _22996;
    int p_22996;
    int  _23002;
    int p_23002;
    int  _23008;
    int p_23008;
    int  _23014;
    int p_23014;
    int  _23020;
    int p_23020;
    int  _23043;
    int p_23043;
    double  sum_23045;
    double psum_23045;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _22990 = blockIdx_x();
    p_22990 = _22990;
    l22988: ;
        _22990 = p_22990;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22996 = blockDim_x();
        p_22996 = _22996;
    l22994: ;
        _22996 = p_22996;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23002 = threadIdx_x();
        p_23002 = _23002;
    l23000: ;
        _23002 = p_23002;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23008 = blockIdx_y();
        p_23008 = _23008;
    l23006: ;
        _23008 = p_23008;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23014 = blockDim_y();
        p_23014 = _23014;
    l23012: ;
        _23014 = p_23014;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23020 = threadIdx_y();
        p_23020 = _23020;
    l23018: ;
        _23020 = p_23020;
        #line 11 "main.impala"
        int _23024;
        _23024 = _20646_22982.e2;
        #line 155 "gpu_device.impala"
        int _23021;
        _23021 = _22990 * _22996;
        #line 155 "gpu_device.impala"
        int gid_x_23022;
        gid_x_23022 = _23021 + _23002;
        #line 160 "gpu_device.impala"
        bool _23025;
        _23025 = gid_x_23022 < _23024;
        #line 160 "gpu_device.impala"
        if (_23025) goto l23026; else goto l23091;
    l23091: ;
        #line 163 "gpu_device.impala"
        goto l23090;
    l23026: ;
        #line 157 "gpu_device.impala"
        int _23027;
        _23027 = _23008 * _23014;
        #line 157 "gpu_device.impala"
        int gid_y_23028;
        gid_y_23028 = _23027 + _23020;
        #line 11 "main.impala"
        int _23030;
        _23030 = _20646_22982.e3;
        #line 160 "gpu_device.impala"
        bool _23031;
        _23031 = gid_y_23028 < _23030;
        #line 160 "gpu_device.impala"
        if (_23031) goto l23032; else goto l23089;
    l23089: ;
        #line 163 "gpu_device.impala"
        goto l23090;
    l23090: ;
        return ;
    l23032: ;
        #line 45 "gpu_device.impala"
        char* _23073;
        _23073 = _20648_22984.e1;
        #line 50 "gpu_device.impala"
        int _23062;
        _23062 = gid_y_23028 * _23024;
        #line 50 "gpu_device.impala"
        struct_Buffer_5425 _23059;
        _23059 = _20646_22982.e1;
        #line 45 "gpu_device.impala"
        double* _23074;
        union { double* dst; char* src; } u_23074;
        u_23074.src = _23073;
        _23074 = u_23074.dst;
        #line 50 "gpu_device.impala"
        int _23075;
        _23075 = _23062 + gid_x_23022;
        #line 45 "gpu_device.impala"
        double* _23076;
        _23076 = _23074 + _23075;
        #line 4 "gaussian.impala"
        int _23034;
        _23034 = _20647_22983.e1;
        #line 50 "gpu_device.impala"
        char* _23060;
        _23060 = _23059.e1;
        #line 4 "gaussian.impala"
        int h_anchor_23036;
        h_anchor_23036 = _23034 / 2;
        #line 50 "gpu_device.impala"
        double* _23061;
        union { double* dst; char* src; } u_23061;
        u_23061.src = _23060;
        _23061 = u_23061.dst;
        #line 17 "gaussian.impala"
        bool _23037;
        _23037 = h_anchor_23036 <= gid_x_23022;
        #line 17 "gaussian.impala"
        if (_23037) goto l23038; else goto l23088;
    l23088: ;
        #line 27 "gaussian.impala"
        goto l23082;
    l23038: ;
        #line 17 "gaussian.impala"
        int _23039;
        _23039 = _23024 - h_anchor_23036;
        #line 17 "gaussian.impala"
        bool _23040;
        _23040 = gid_x_23022 < _23039;
        #line 17 "gaussian.impala"
        if (_23040) goto l23041; else goto l23081;
    l23081: ;
        #line 27 "gaussian.impala"
        goto l23082;
    l23082: ;
        #line 50 "gpu_device.impala"
        double* _23083;
        _23083 = _23061 + _23075;
        #line 50 "gpu_device.impala"
        double _23084;
        _23084 = *_23083;
        #line 50 "gpu_device.impala"
        double _23086;
        _23086 = _23084;
        #line 45 "gpu_device.impala"
        *_23076 = _23086;
        return ;
    l23041: ;
        #line 19 "gaussian.impala"
        int _23047;
        _23047 = 1 + h_anchor_23036;
        #line 55 "gpu_device.impala"
        struct_Buffer_5425 _23052;
        _23052 = _20647_22983.e0;
        #line 19 "gaussian.impala"
        int _23079;
        _23079 = 0 - h_anchor_23036;
        #line 55 "gpu_device.impala"
        char* _23053;
        _23053 = _23052.e1;
        #line 55 "gpu_device.impala"
        double* _23054;
        union { double* dst; char* src; } u_23054;
        u_23054.src = _23053;
        _23054 = u_23054.dst;
        #line 19 "gpu_device.impala"
        p_23043 = _23079;
        psum_23045 = 0.000000e+00;
        goto l23042;
    l23042: ;
        _23043 = p_23043;
        sum_23045 = psum_23045;
        #line 19 "gpu_device.impala"
        bool _23048;
        _23048 = _23043 < _23047;
        #line 19 "gpu_device.impala"
        if (_23048) goto l23049; else goto l23072;
    l23072: ;
        #line 45 "gpu_device.impala"
        *_23076 = sum_23045;
        return ;
    l23049: ;
        #line 23 "gpu_device.impala"
        int _23050;
        _23050 = 1 + _23043;
        #line 21 "gaussian.impala"
        int _23055;
        _23055 = _23043 + h_anchor_23036;
        #line 21 "gaussian.impala"
        int _23063;
        _23063 = gid_x_23022 + _23043;
        #line 54 "gpu_device.impala"
        double* i_23056;
        i_23056 = _23054 + _23055;
        #line 50 "gpu_device.impala"
        int _23064;
        _23064 = _23062 + _23063;
        #line 55 "gpu_device.impala"
        double _23057;
        _23057 = *i_23056;
        #line 50 "gpu_device.impala"
        double* _23065;
        _23065 = _23061 + _23064;
        #line 55 "gpu_device.impala"
        double _23068;
        _23068 = _23057;
        #line 50 "gpu_device.impala"
        double _23066;
        _23066 = *_23065;
        #line 50 "gpu_device.impala"
        double _23069;
        _23069 = _23066;
        #line 21 "gaussian.impala"
        double _23070;
        _23070 = _23068 * _23069;
        #line 21 "gaussian.impala"
        double _23071;
        _23071 = sum_23045 + _23070;
        #line 19 "gpu_device.impala"
        p_23043 = _23050;
        psum_23045 = _23071;
        goto l23042;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_20775(struct_filter_5428 _20778_23095, struct_image_5424 _20779_23096, struct_Buffer_5425 _20780_23097, double* _20781_23098, struct_Buffer_5425 _20782_23099) {
    __shared__ double ds_img[134][7];
    int  _23102;
    int p_23102;
    int  _23105;
    int p_23105;
    int  _23108;
    int p_23108;
    int  _23111;
    int p_23111;
    int  _23114;
    int p_23114;
    int  _23117;
    int p_23117;
    int  _23136;
    int p_23136;
    double  sum_23138;
    double psum_23138;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _23102 = blockIdx_x();
    p_23102 = _23102;
    l23100: ;
        _23102 = p_23102;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23105 = blockDim_x();
        p_23105 = _23105;
    l23103: ;
        _23105 = p_23105;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23108 = threadIdx_x();
        p_23108 = _23108;
    l23106: ;
        _23108 = p_23108;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23111 = blockIdx_y();
        p_23111 = _23111;
    l23109: ;
        _23111 = p_23111;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23114 = blockDim_y();
        p_23114 = _23114;
    l23112: ;
        _23114 = p_23114;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23117 = threadIdx_y();
        p_23117 = _23117;
    l23115: ;
        _23117 = p_23117;
        #line 155 "gpu_device.impala"
        int _23118;
        _23118 = _23102 * _23105;
        #line 155 "gpu_device.impala"
        int gid_x_23119;
        gid_x_23119 = _23118 + _23108;
        #line 11 "main.impala"
        int _23120;
        _23120 = _20779_23096.e2;
        #line 160 "gpu_device.impala"
        bool _23121;
        _23121 = gid_x_23119 < _23120;
        #line 160 "gpu_device.impala"
        if (_23121) goto l23122; else goto l23180;
    l23180: ;
        #line 163 "gpu_device.impala"
        goto l23179;
    l23122: ;
        #line 157 "gpu_device.impala"
        int _23123;
        _23123 = _23111 * _23114;
        #line 11 "main.impala"
        int _23125;
        _23125 = _20779_23096.e3;
        #line 157 "gpu_device.impala"
        int gid_y_23124;
        gid_y_23124 = _23123 + _23117;
        #line 160 "gpu_device.impala"
        bool _23126;
        _23126 = gid_y_23124 < _23125;
        #line 160 "gpu_device.impala"
        if (_23126) goto l23127; else goto l23178;
    l23178: ;
        #line 163 "gpu_device.impala"
        goto l23179;
    l23179: ;
        return ;
    l23127: ;
        #line 45 "gpu_device.impala"
        char* _23163;
        _23163 = _20780_23097.e1;
        #line 50 "gpu_device.impala"
        char* _23150;
        _23150 = _20782_23099.e1;
        #line 50 "gpu_device.impala"
        int _23165;
        _23165 = gid_y_23124 * _23120;
        #line 6 "gaussian.impala"
        int _23128;
        _23128 = _20778_23095.e2;
        #line 45 "gpu_device.impala"
        double* _23164;
        union { double* dst; char* src; } u_23164;
        u_23164.src = _23163;
        _23164 = u_23164.dst;
        #line 50 "gpu_device.impala"
        double* _23151;
        union { double* dst; char* src; } u_23151;
        u_23151.src = _23150;
        _23151 = u_23151.dst;
        #line 50 "gpu_device.impala"
        int _23166;
        _23166 = _23165 + gid_x_23119;
        #line 6 "gaussian.impala"
        int v_anchor_23129;
        v_anchor_23129 = _23128 / 2;
        #line 45 "gpu_device.impala"
        double* _23167;
        _23167 = _23164 + _23166;
        #line 39 "gaussian.impala"
        bool _23130;
        _23130 = v_anchor_23129 <= gid_y_23124;
        #line 39 "gaussian.impala"
        if (_23130) goto l23131; else goto l23177;
    l23177: ;
        #line 49 "gaussian.impala"
        goto l23171;
    l23131: ;
        #line 39 "gaussian.impala"
        int _23132;
        _23132 = _23125 - v_anchor_23129;
        #line 39 "gaussian.impala"
        bool _23133;
        _23133 = gid_y_23124 < _23132;
        #line 39 "gaussian.impala"
        if (_23133) goto l23134; else goto l23170;
    l23170: ;
        #line 49 "gaussian.impala"
        goto l23171;
    l23171: ;
        #line 50 "gpu_device.impala"
        double* _23172;
        _23172 = _23151 + _23166;
        #line 50 "gpu_device.impala"
        double _23173;
        _23173 = *_23172;
        #line 50 "gpu_device.impala"
        double _23175;
        _23175 = _23173;
        #line 45 "gpu_device.impala"
        *_23167 = _23175;
        return ;
    l23134: ;
        #line 41 "gaussian.impala"
        int _23139;
        _23139 = 1 + v_anchor_23129;
        #line 55 "gpu_device.impala"
        struct_Buffer_5425 _23143;
        _23143 = _20778_23095.e0;
        #line 41 "gaussian.impala"
        int _23169;
        _23169 = 0 - v_anchor_23129;
        #line 55 "gpu_device.impala"
        char* _23144;
        _23144 = _23143.e1;
        #line 55 "gpu_device.impala"
        double* _23145;
        union { double* dst; char* src; } u_23145;
        u_23145.src = _23144;
        _23145 = u_23145.dst;
        #line 19 "gpu_device.impala"
        p_23136 = _23169;
        psum_23138 = 0.000000e+00;
        goto l23135;
    l23135: ;
        _23136 = p_23136;
        sum_23138 = psum_23138;
        #line 19 "gpu_device.impala"
        bool _23140;
        _23140 = _23136 < _23139;
        #line 19 "gpu_device.impala"
        if (_23140) goto l23141; else goto l23162;
    l23162: ;
        #line 45 "gpu_device.impala"
        *_23167 = sum_23138;
        return ;
    l23141: ;
        #line 43 "gaussian.impala"
        int _23146;
        _23146 = _23136 + v_anchor_23129;
        #line 43 "gaussian.impala"
        int _23152;
        _23152 = gid_y_23124 + _23136;
        #line 50 "gpu_device.impala"
        int _23153;
        _23153 = _23152 * _23120;
        #line 23 "gpu_device.impala"
        int _23142;
        _23142 = 1 + _23136;
        #line 54 "gpu_device.impala"
        double* i_23147;
        i_23147 = _23145 + _23146;
        #line 55 "gpu_device.impala"
        double _23148;
        _23148 = *i_23147;
        #line 50 "gpu_device.impala"
        int _23154;
        _23154 = _23153 + gid_x_23119;
        #line 55 "gpu_device.impala"
        double _23158;
        _23158 = _23148;
        #line 50 "gpu_device.impala"
        double* _23155;
        _23155 = _23151 + _23154;
        #line 50 "gpu_device.impala"
        double _23156;
        _23156 = *_23155;
        #line 50 "gpu_device.impala"
        double _23159;
        _23159 = _23156;
        #line 43 "gaussian.impala"
        double _23160;
        _23160 = _23158 * _23159;
        #line 43 "gaussian.impala"
        double _23161;
        _23161 = sum_23138 + _23160;
        #line 19 "gpu_device.impala"
        p_23136 = _23142;
        psum_23138 = _23161;
        goto l23135;
}

}
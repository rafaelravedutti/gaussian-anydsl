
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    float e[4];
} array_11173;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_40799(float*, float*);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_40799(float* _40802_44104, float* _40803_44105) {
    int  threadIdx_x_44114;
    int pthreadIdx_x_44114;
    int  blockDim_x_44120;
    int pblockDim_x_44120;
    int  blockIdx_x_44126;
    int pblockIdx_x_44126;
    int  threadIdx_y_44132;
    int pthreadIdx_y_44132;
    int  blockDim_y_44138;
    int pblockDim_y_44138;
    int  blockIdx_y_44144;
    int pblockIdx_y_44144;
    int  converge_44153;
    int pconverge_44153;
    int  converge_44158;
    int pconverge_44158;
    float  _44170;
    float p_44170;
    int  converge_44176;
    int pconverge_44176;
    int  converge_44180;
    int pconverge_44180;
    float  _44190;
    float p_44190;
    int  converge_44194;
    int pconverge_44194;
    int  converge_44198;
    int pconverge_44198;
    float  _44206;
    float p_44206;
    int  converge_44211;
    int pconverge_44211;
    int  converge_44215;
    int pconverge_44215;
    float  _44223;
    float p_44223;
    int  lower_44274;
    int plower_44274;
    int  upper_44275;
    int pupper_44275;
    int  step_44276;
    int pstep_44276;
    
    array_11173 tmp_44183_slot;
    array_11173* tmp_44183;
    tmp_44183 = &tmp_44183_slot;
    threadIdx_x_44114 = threadIdx_x();
    pthreadIdx_x_44114 = threadIdx_x_44114;
    l44112: ;
        threadIdx_x_44114 = pthreadIdx_x_44114;
        blockDim_x_44120 = blockDim_x();
        pblockDim_x_44120 = blockDim_x_44120;
    l44118: ;
        blockDim_x_44120 = pblockDim_x_44120;
        blockIdx_x_44126 = blockIdx_x();
        pblockIdx_x_44126 = blockIdx_x_44126;
    l44124: ;
        blockIdx_x_44126 = pblockIdx_x_44126;
        threadIdx_y_44132 = threadIdx_y();
        pthreadIdx_y_44132 = threadIdx_y_44132;
    l44130: ;
        threadIdx_y_44132 = pthreadIdx_y_44132;
        blockDim_y_44138 = blockDim_y();
        pblockDim_y_44138 = blockDim_y_44138;
    l44136: ;
        blockDim_y_44138 = pblockDim_y_44138;
        blockIdx_y_44144 = blockIdx_y();
        pblockIdx_y_44144 = blockIdx_y_44144;
    l44142: ;
        blockIdx_y_44144 = pblockIdx_y_44144;
        int _44146;
        _44146 = blockDim_x_44120 * blockIdx_x_44126;
        int _44147;
        _44147 = threadIdx_x_44114 + _44146;
        int _44148;
        _44148 = -2 + _44147;
        bool _44150;
        _44150 = _44148 < 0;
        if (_44150) goto l44151; else goto l44367;
    l44367: ;
        pconverge_44153 = _44148;
        goto l44152;
    l44151: ;
        pconverge_44153 = 0;
        goto l44152;
    l44152: ;
        converge_44153 = pconverge_44153;
        bool _44155;
        _44155 = 4096 <= converge_44153;
        if (_44155) goto l44156; else goto l44366;
    l44366: ;
        pconverge_44158 = converge_44153;
        goto l44157;
    l44156: ;
        pconverge_44158 = 4095;
        goto l44157;
    l44157: ;
        converge_44158 = pconverge_44158;
        int _44163;
        _44163 = blockDim_y_44138 * blockIdx_y_44144;
        int _44164;
        _44164 = threadIdx_y_44132 + _44163;
        int _44165;
        _44165 = 4096 * _44164;
        int _44166;
        _44166 = _44165 + converge_44158;
        float* idx_44167;
        idx_44167 = _40803_44105 + _44166;
        _44170 = __ldg(idx_44167);
        p_44170 = _44170;
    l44168: ;
        _44170 = p_44170;
        int _44172;
        _44172 = -1 + _44147;
        bool _44173;
        _44173 = _44172 < 0;
        if (_44173) goto l44174; else goto l44365;
    l44365: ;
        pconverge_44176 = _44172;
        goto l44175;
    l44174: ;
        pconverge_44176 = 0;
        goto l44175;
    l44175: ;
        converge_44176 = pconverge_44176;
        bool _44177;
        _44177 = 4096 <= converge_44176;
        if (_44177) goto l44178; else goto l44364;
    l44364: ;
        pconverge_44180 = converge_44176;
        goto l44179;
    l44178: ;
        pconverge_44180 = 4095;
        goto l44179;
    l44179: ;
        converge_44180 = pconverge_44180;
        int _44186;
        _44186 = _44165 + converge_44180;
        float* idx_44187;
        idx_44187 = _40803_44105 + _44186;
        float* _44184;
        _44184 = &tmp_44183->e[0];
        *_44184 = _44170;
        _44190 = __ldg(idx_44187);
        p_44190 = _44190;
    l44188: ;
        _44190 = p_44190;
        bool _44191;
        _44191 = _44147 < 0;
        if (_44191) goto l44192; else goto l44363;
    l44363: ;
        pconverge_44194 = _44147;
        goto l44193;
    l44192: ;
        pconverge_44194 = 0;
        goto l44193;
    l44193: ;
        converge_44194 = pconverge_44194;
        bool _44195;
        _44195 = 4096 <= converge_44194;
        if (_44195) goto l44196; else goto l44362;
    l44362: ;
        pconverge_44198 = converge_44194;
        goto l44197;
    l44196: ;
        pconverge_44198 = 4095;
        goto l44197;
    l44197: ;
        converge_44198 = pconverge_44198;
        int _44202;
        _44202 = _44165 + converge_44198;
        float* idx_44203;
        idx_44203 = _40803_44105 + _44202;
        float* unroll_44200;
        unroll_44200 = &tmp_44183->e[1];
        *unroll_44200 = _44190;
        _44206 = __ldg(idx_44203);
        p_44206 = _44206;
    l44204: ;
        _44206 = p_44206;
        int _44207;
        _44207 = 1 + _44147;
        bool _44208;
        _44208 = _44207 < 0;
        if (_44208) goto l44209; else goto l44361;
    l44361: ;
        pconverge_44211 = _44207;
        goto l44210;
    l44209: ;
        pconverge_44211 = 0;
        goto l44210;
    l44210: ;
        converge_44211 = pconverge_44211;
        bool _44212;
        _44212 = 4096 <= converge_44211;
        if (_44212) goto l44213; else goto l44360;
    l44360: ;
        pconverge_44215 = converge_44211;
        goto l44214;
    l44213: ;
        pconverge_44215 = 4095;
        goto l44214;
    l44214: ;
        converge_44215 = pconverge_44215;
        int _44219;
        _44219 = _44165 + converge_44215;
        float* _44217;
        _44217 = &tmp_44183->e[2];
        float* idx_44220;
        idx_44220 = _40803_44105 + _44219;
        *_44217 = _44206;
        _44223 = __ldg(idx_44220);
        p_44223 = _44223;
    l44221: ;
        _44223 = p_44223;
        float* _44225;
        _44225 = &tmp_44183->e[3];
        *_44225 = _44223;
        float _44227;
        _44227 = *_44184;
        float _44231;
        _44231 = _44227;
        float _44229;
        _44229 = *unroll_44200;
        float _44230;
        _44230 = _44229;
        bool _44232;
        _44232 = _44230 < _44231;
        if (_44232) goto l44233; else goto l44358;
    l44358: ;
        goto l44234;
    l44233: ;
        float _44350;
        _44350 = *_44184;
        float _44356;
        _44356 = _44350;
        float _44352;
        _44352 = *unroll_44200;
        float _44354;
        _44354 = _44352;
        *_44184 = _44354;
        *unroll_44200 = _44356;
        goto l44234;
    l44234: ;
        float _44236;
        _44236 = *_44217;
        float _44240;
        _44240 = _44236;
        float _44238;
        _44238 = *_44225;
        float _44239;
        _44239 = _44238;
        bool _44241;
        _44241 = _44239 < _44240;
        if (_44241) goto l44242; else goto l44348;
    l44348: ;
        goto l44243;
    l44242: ;
        float _44340;
        _44340 = *_44217;
        float _44346;
        _44346 = _44340;
        float _44342;
        _44342 = *_44225;
        float _44344;
        _44344 = _44342;
        *_44217 = _44344;
        *_44225 = _44346;
        goto l44243;
    l44243: ;
        float _44245;
        _44245 = *_44184;
        float _44249;
        _44249 = _44245;
        float _44247;
        _44247 = *_44217;
        float _44248;
        _44248 = _44247;
        bool _44250;
        _44250 = _44248 < _44249;
        if (_44250) goto l44251; else goto l44338;
    l44338: ;
        goto l44252;
    l44251: ;
        float _44330;
        _44330 = *_44184;
        float _44332;
        _44332 = *_44217;
        float _44336;
        _44336 = _44330;
        float _44334;
        _44334 = _44332;
        *_44184 = _44334;
        *_44217 = _44336;
        goto l44252;
    l44252: ;
        float _44254;
        _44254 = *unroll_44200;
        float _44258;
        _44258 = _44254;
        float _44256;
        _44256 = *_44225;
        float _44257;
        _44257 = _44256;
        bool _44259;
        _44259 = _44257 < _44258;
        if (_44259) goto l44260; else goto l44328;
    l44328: ;
        goto l44261;
    l44260: ;
        float _44320;
        _44320 = *unroll_44200;
        float _44326;
        _44326 = _44320;
        float _44322;
        _44322 = *_44225;
        float _44324;
        _44324 = _44322;
        *unroll_44200 = _44324;
        *_44225 = _44326;
        goto l44261;
    l44261: ;
        float _44263;
        _44263 = *unroll_44200;
        float _44267;
        _44267 = _44263;
        float _44265;
        _44265 = *_44217;
        float _44266;
        _44266 = _44265;
        bool _44268;
        _44268 = _44266 < _44267;
        if (_44268) goto l44269; else goto l44318;
    l44318: ;
        goto l44270;
    l44269: ;
        float _44310;
        _44310 = *unroll_44200;
        float _44316;
        _44316 = _44310;
        float _44312;
        _44312 = *_44217;
        float _44314;
        _44314 = _44312;
        *unroll_44200 = _44314;
        *_44217 = _44316;
        goto l44270;
    l44270: ;
        plower_44274 = 3;
        pupper_44275 = 3;
        pstep_44276 = 2;
        goto l44272;
    l44272: ;
        lower_44274 = plower_44274;
        upper_44275 = pupper_44275;
        step_44276 = pstep_44276;
        bool _44277;
        _44277 = lower_44274 < upper_44275;
        if (_44277) goto l44278; else goto l44302;
    l44302: ;
        int _44305;
        _44305 = _44165 + _44147;
        float _44303;
        _44303 = *unroll_44200;
        float* idx_44306;
        idx_44306 = _40802_44104 + _44305;
        float _44307;
        _44307 = _44303;
        *idx_44306 = _44307;
        return ;
    l44278: ;
        int _44282;
        _44282 = 1 + lower_44274;
        float* idx_44279;
        idx_44279 = &tmp_44183->e[lower_44274];
        float _44280;
        _44280 = *idx_44279;
        float* idx_44283;
        idx_44283 = &tmp_44183->e[_44282];
        float _44286;
        _44286 = _44280;
        float _44284;
        _44284 = *idx_44283;
        float _44285;
        _44285 = _44284;
        bool _44287;
        _44287 = _44285 < _44286;
        if (_44287) goto l44288; else goto l44301;
    l44301: ;
        goto l44289;
    l44288: ;
        float _44293;
        _44293 = *idx_44279;
        float _44299;
        _44299 = _44293;
        float _44295;
        _44295 = *idx_44283;
        float _44297;
        _44297 = _44295;
        *idx_44279 = _44297;
        *idx_44283 = _44299;
        goto l44289;
    l44289: ;
        int _44291;
        _44291 = lower_44274 + step_44276;
        plower_44274 = _44291;
        pupper_44275 = upper_44275;
        pstep_44276 = step_44276;
        goto l44272;
}

}

#include <hip/hip_runtime.h>
extern "C" {
__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_35652(float*, float*);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_35652(float* _35655_38561, float* _35656_38562) {
    int  threadIdx_x_38568;
    int pthreadIdx_x_38568;
    int  blockDim_x_38574;
    int pblockDim_x_38574;
    int  blockIdx_x_38580;
    int pblockIdx_x_38580;
    int  _38586;
    int p_38586;
    int  _38592;
    int p_38592;
    int  _38598;
    int p_38598;
    int  _38601;
    int p_38601;
    int  converge_38608;
    int pconverge_38608;
    int  converge_38613;
    int pconverge_38613;
    int  converge_38621;
    int pconverge_38621;
    int  converge_38625;
    int pconverge_38625;
    float  _38635;
    float p_38635;
    int  converge_38640;
    int pconverge_38640;
    int  converge_38644;
    int pconverge_38644;
    int  converge_38648;
    int pconverge_38648;
    int  converge_38652;
    int pconverge_38652;
    float  _38658;
    float p_38658;
    int  converge_38661;
    int pconverge_38661;
    int  converge_38665;
    int pconverge_38665;
    int  converge_38668;
    int pconverge_38668;
    int  converge_38672;
    int pconverge_38672;
    float  _38678;
    float p_38678;
    int  converge_38684;
    int pconverge_38684;
    int  converge_38688;
    int pconverge_38688;
    int  converge_38691;
    int pconverge_38691;
    int  converge_38695;
    int pconverge_38695;
    float  _38701;
    float p_38701;
    int  converge_38704;
    int pconverge_38704;
    int  converge_38708;
    int pconverge_38708;
    int  converge_38713;
    int pconverge_38713;
    int  converge_38717;
    int pconverge_38717;
    float  _38723;
    float p_38723;
    threadIdx_x_38568 = threadIdx_x();
    pthreadIdx_x_38568 = threadIdx_x_38568;
    l38566: ;
        threadIdx_x_38568 = pthreadIdx_x_38568;
        blockDim_x_38574 = blockDim_x();
        pblockDim_x_38574 = blockDim_x_38574;
    l38572: ;
        blockDim_x_38574 = pblockDim_x_38574;
        blockIdx_x_38580 = blockIdx_x();
        pblockIdx_x_38580 = blockIdx_x_38580;
    l38578: ;
        blockIdx_x_38580 = pblockIdx_x_38580;
        _38586 = threadIdx_y();
        p_38586 = _38586;
    l38584: ;
        _38586 = p_38586;
        _38592 = blockDim_y();
        p_38592 = _38592;
    l38590: ;
        _38592 = p_38592;
        _38598 = blockIdx_y();
        p_38598 = _38598;
    l38596: ;
        _38598 = p_38598;
        _38601 = blockDim_y();
        p_38601 = _38601;
    l38599: ;
        _38601 = p_38601;
        int _38602;
        _38602 = blockDim_x_38574 * blockIdx_x_38580;
        int _38603;
        _38603 = threadIdx_x_38568 + _38602;
        bool _38605;
        _38605 = _38603 < 0;
        if (_38605) goto l38606; else goto l38761;
    l38761: ;
        pconverge_38608 = _38603;
        goto l38607;
    l38606: ;
        pconverge_38608 = 0;
        goto l38607;
    l38607: ;
        converge_38608 = pconverge_38608;
        bool _38610;
        _38610 = 2048 <= converge_38608;
        if (_38610) goto l38611; else goto l38760;
    l38760: ;
        pconverge_38613 = converge_38608;
        goto l38612;
    l38611: ;
        pconverge_38613 = 2047;
        goto l38612;
    l38612: ;
        converge_38613 = pconverge_38613;
        int _38615;
        _38615 = _38592 * _38598;
        int gid_y_38616;
        gid_y_38616 = _38586 + _38615;
        int _38617;
        _38617 = -1 + gid_y_38616;
        bool _38618;
        _38618 = _38617 < 0;
        if (_38618) goto l38619; else goto l38759;
    l38759: ;
        pconverge_38621 = _38617;
        goto l38620;
    l38619: ;
        pconverge_38621 = 0;
        goto l38620;
    l38620: ;
        converge_38621 = pconverge_38621;
        bool _38622;
        _38622 = 2048 <= converge_38621;
        if (_38622) goto l38623; else goto l38758;
    l38758: ;
        pconverge_38625 = converge_38621;
        goto l38624;
    l38623: ;
        pconverge_38625 = 2047;
        goto l38624;
    l38624: ;
        converge_38625 = pconverge_38625;
        int _38630;
        _38630 = 2048 * converge_38625;
        int _38631;
        _38631 = _38630 + converge_38613;
        float* idx_38632;
        idx_38632 = _35655_38561 + _38631;
        _38635 = __ldg(idx_38632);
        p_38635 = _38635;
    l38633: ;
        _38635 = p_38635;
        int _38636;
        _38636 = -1 + _38603;
        bool _38637;
        _38637 = _38636 < 0;
        if (_38637) goto l38638; else goto l38757;
    l38757: ;
        pconverge_38640 = _38636;
        goto l38639;
    l38638: ;
        pconverge_38640 = 0;
        goto l38639;
    l38639: ;
        converge_38640 = pconverge_38640;
        bool _38641;
        _38641 = 2048 <= converge_38640;
        if (_38641) goto l38642; else goto l38756;
    l38756: ;
        pconverge_38644 = converge_38640;
        goto l38643;
    l38642: ;
        pconverge_38644 = 2047;
        goto l38643;
    l38643: ;
        converge_38644 = pconverge_38644;
        bool _38645;
        _38645 = gid_y_38616 < 0;
        if (_38645) goto l38646; else goto l38755;
    l38755: ;
        pconverge_38648 = gid_y_38616;
        goto l38647;
    l38646: ;
        pconverge_38648 = 0;
        goto l38647;
    l38647: ;
        converge_38648 = pconverge_38648;
        bool _38649;
        _38649 = 2048 <= converge_38648;
        if (_38649) goto l38650; else goto l38754;
    l38754: ;
        pconverge_38652 = converge_38648;
        goto l38651;
    l38650: ;
        pconverge_38652 = 2047;
        goto l38651;
    l38651: ;
        converge_38652 = pconverge_38652;
        int _38653;
        _38653 = 2048 * converge_38652;
        int _38654;
        _38654 = _38653 + converge_38644;
        float* idx_38655;
        idx_38655 = _35655_38561 + _38654;
        _38658 = __ldg(idx_38655);
        p_38658 = _38658;
    l38656: ;
        _38658 = p_38658;
        if (_38605) goto l38659; else goto l38753;
    l38753: ;
        pconverge_38661 = _38603;
        goto l38660;
    l38659: ;
        pconverge_38661 = 0;
        goto l38660;
    l38660: ;
        converge_38661 = pconverge_38661;
        bool _38662;
        _38662 = 2048 <= converge_38661;
        if (_38662) goto l38663; else goto l38752;
    l38752: ;
        pconverge_38665 = converge_38661;
        goto l38664;
    l38663: ;
        pconverge_38665 = 2047;
        goto l38664;
    l38664: ;
        converge_38665 = pconverge_38665;
        if (_38645) goto l38666; else goto l38751;
    l38751: ;
        pconverge_38668 = gid_y_38616;
        goto l38667;
    l38666: ;
        pconverge_38668 = 0;
        goto l38667;
    l38667: ;
        converge_38668 = pconverge_38668;
        bool _38669;
        _38669 = 2048 <= converge_38668;
        if (_38669) goto l38670; else goto l38750;
    l38750: ;
        pconverge_38672 = converge_38668;
        goto l38671;
    l38670: ;
        pconverge_38672 = 2047;
        goto l38671;
    l38671: ;
        converge_38672 = pconverge_38672;
        int _38673;
        _38673 = 2048 * converge_38672;
        int _38674;
        _38674 = _38673 + converge_38665;
        float* idx_38675;
        idx_38675 = _35655_38561 + _38674;
        _38678 = __ldg(idx_38675);
        p_38678 = _38678;
    l38676: ;
        _38678 = p_38678;
        int _38680;
        _38680 = 1 + _38603;
        bool _38681;
        _38681 = _38680 < 0;
        if (_38681) goto l38682; else goto l38749;
    l38749: ;
        pconverge_38684 = _38680;
        goto l38683;
    l38682: ;
        pconverge_38684 = 0;
        goto l38683;
    l38683: ;
        converge_38684 = pconverge_38684;
        bool _38685;
        _38685 = 2048 <= converge_38684;
        if (_38685) goto l38686; else goto l38748;
    l38748: ;
        pconverge_38688 = converge_38684;
        goto l38687;
    l38686: ;
        pconverge_38688 = 2047;
        goto l38687;
    l38687: ;
        converge_38688 = pconverge_38688;
        if (_38645) goto l38689; else goto l38747;
    l38747: ;
        pconverge_38691 = gid_y_38616;
        goto l38690;
    l38689: ;
        pconverge_38691 = 0;
        goto l38690;
    l38690: ;
        converge_38691 = pconverge_38691;
        bool _38692;
        _38692 = 2048 <= converge_38691;
        if (_38692) goto l38693; else goto l38746;
    l38746: ;
        pconverge_38695 = converge_38691;
        goto l38694;
    l38693: ;
        pconverge_38695 = 2047;
        goto l38694;
    l38694: ;
        converge_38695 = pconverge_38695;
        int _38696;
        _38696 = 2048 * converge_38695;
        int _38697;
        _38697 = _38696 + converge_38688;
        float* idx_38698;
        idx_38698 = _35655_38561 + _38697;
        _38701 = __ldg(idx_38698);
        p_38701 = _38701;
    l38699: ;
        _38701 = p_38701;
        if (_38605) goto l38702; else goto l38745;
    l38745: ;
        pconverge_38704 = _38603;
        goto l38703;
    l38702: ;
        pconverge_38704 = 0;
        goto l38703;
    l38703: ;
        converge_38704 = pconverge_38704;
        bool _38705;
        _38705 = 2048 <= converge_38704;
        if (_38705) goto l38706; else goto l38744;
    l38744: ;
        pconverge_38708 = converge_38704;
        goto l38707;
    l38706: ;
        pconverge_38708 = 2047;
        goto l38707;
    l38707: ;
        converge_38708 = pconverge_38708;
        int _38709;
        _38709 = 1 + gid_y_38616;
        bool _38710;
        _38710 = _38709 < 0;
        if (_38710) goto l38711; else goto l38743;
    l38743: ;
        pconverge_38713 = _38709;
        goto l38712;
    l38711: ;
        pconverge_38713 = 0;
        goto l38712;
    l38712: ;
        converge_38713 = pconverge_38713;
        bool _38714;
        _38714 = 2048 <= converge_38713;
        if (_38714) goto l38715; else goto l38742;
    l38742: ;
        pconverge_38717 = converge_38713;
        goto l38716;
    l38715: ;
        pconverge_38717 = 2047;
        goto l38716;
    l38716: ;
        converge_38717 = pconverge_38717;
        int _38718;
        _38718 = 2048 * converge_38717;
        int _38719;
        _38719 = _38718 + converge_38708;
        float* idx_38720;
        idx_38720 = _35655_38561 + _38719;
        _38723 = __ldg(idx_38720);
        p_38723 = _38723;
    l38721: ;
        _38723 = p_38723;
        float _38731;
        _38731 = 2.000000e-01f * _38658;
        int _38724;
        _38724 = 2048 * gid_y_38616;
        float _38738;
        _38738 = 2.000000e-01f * _38723;
        float _38734;
        _38734 = 2.000000e-01f * _38678;
        float _38736;
        _38736 = 2.000000e-01f * _38701;
        float _38729;
        _38729 = 2.000000e-01f * _38635;
        int _38725;
        _38725 = _38724 + _38603;
        float _38730;
        _38730 = 0.000000e+00f + _38729;
        float* idx_38726;
        idx_38726 = _35656_38562 + _38725;
        float _38732;
        _38732 = _38730 + _38731;
        float _38735;
        _38735 = _38732 + _38734;
        float _38737;
        _38737 = _38735 + _38736;
        float _38739;
        _38739 = _38737 + _38738;
        *idx_38726 = _38739;
        return ;
}

}
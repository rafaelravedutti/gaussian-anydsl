
#include <hip/hip_runtime.h>
extern "C" {
__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_14349(float*, float*);
__global__ void lambda_14462(float*, float*);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_14349(float* _14352_15506, float* _14353_15507) {
    int  threadIdx_x_15513;
    int pthreadIdx_x_15513;
    int  blockDim_x_15519;
    int pblockDim_x_15519;
    int  blockIdx_x_15525;
    int pblockIdx_x_15525;
    int  threadIdx_y_15531;
    int pthreadIdx_y_15531;
    int  blockDim_y_15537;
    int pblockDim_y_15537;
    int  blockIdx_y_15543;
    int pblockIdx_y_15543;
    threadIdx_x_15513 = threadIdx_x();
    pthreadIdx_x_15513 = threadIdx_x_15513;
    l15511: ;
        threadIdx_x_15513 = pthreadIdx_x_15513;
        blockDim_x_15519 = blockDim_x();
        pblockDim_x_15519 = blockDim_x_15519;
    l15517: ;
        blockDim_x_15519 = pblockDim_x_15519;
        blockIdx_x_15525 = blockIdx_x();
        pblockIdx_x_15525 = blockIdx_x_15525;
    l15523: ;
        blockIdx_x_15525 = pblockIdx_x_15525;
        threadIdx_y_15531 = threadIdx_y();
        pthreadIdx_y_15531 = threadIdx_y_15531;
    l15529: ;
        threadIdx_y_15531 = pthreadIdx_y_15531;
        blockDim_y_15537 = blockDim_y();
        pblockDim_y_15537 = blockDim_y_15537;
    l15535: ;
        blockDim_y_15537 = pblockDim_y_15537;
        blockIdx_y_15543 = blockIdx_y();
        pblockIdx_y_15543 = blockIdx_y_15543;
    l15541: ;
        blockIdx_y_15543 = pblockIdx_y_15543;
        int _15548;
        _15548 = blockDim_x_15519 * blockIdx_x_15525;
        int _15545;
        _15545 = blockDim_y_15537 * blockIdx_y_15543;
        int _15549;
        _15549 = threadIdx_x_15513 + _15548;
        int _15546;
        _15546 = threadIdx_y_15531 + _15545;
        int _15547;
        _15547 = 4096 * _15546;
        int _15550;
        _15550 = _15547 + _15549;
        float* _15555;
        _15555 = _14352_15506 + _15550;
        float* _15551;
        _15551 = _14353_15507 + _15550;
        float _15552;
        _15552 = *_15551;
        float _15557;
        _15557 = _15552;
        *_15555 = _15557;
        return ;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_14462(float* _14465_15562, float* _14466_15563) {
    int  threadIdx_x_15566;
    int pthreadIdx_x_15566;
    int  blockDim_x_15569;
    int pblockDim_x_15569;
    int  blockIdx_x_15572;
    int pblockIdx_x_15572;
    int  threadIdx_y_15575;
    int pthreadIdx_y_15575;
    int  blockDim_y_15578;
    int pblockDim_y_15578;
    int  blockIdx_y_15581;
    int pblockIdx_y_15581;
    threadIdx_x_15566 = threadIdx_x();
    pthreadIdx_x_15566 = threadIdx_x_15566;
    l15564: ;
        threadIdx_x_15566 = pthreadIdx_x_15566;
        blockDim_x_15569 = blockDim_x();
        pblockDim_x_15569 = blockDim_x_15569;
    l15567: ;
        blockDim_x_15569 = pblockDim_x_15569;
        blockIdx_x_15572 = blockIdx_x();
        pblockIdx_x_15572 = blockIdx_x_15572;
    l15570: ;
        blockIdx_x_15572 = pblockIdx_x_15572;
        threadIdx_y_15575 = threadIdx_y();
        pthreadIdx_y_15575 = threadIdx_y_15575;
    l15573: ;
        threadIdx_y_15575 = pthreadIdx_y_15575;
        blockDim_y_15578 = blockDim_y();
        pblockDim_y_15578 = blockDim_y_15578;
    l15576: ;
        blockDim_y_15578 = pblockDim_y_15578;
        blockIdx_y_15581 = blockIdx_y();
        pblockIdx_y_15581 = blockIdx_y_15581;
    l15579: ;
        blockIdx_y_15581 = pblockIdx_y_15581;
        int _15585;
        _15585 = blockDim_x_15569 * blockIdx_x_15572;
        int _15586;
        _15586 = threadIdx_x_15566 + _15585;
        int _15582;
        _15582 = blockDim_y_15578 * blockIdx_y_15581;
        int _15583;
        _15583 = threadIdx_y_15575 + _15582;
        int _15584;
        _15584 = 4096 * _15583;
        int _15587;
        _15587 = _15584 + _15586;
        float* _15591;
        _15591 = _14466_15563 + _15587;
        float* _15588;
        _15588 = _14465_15562 + _15587;
        float _15589;
        _15589 = *_15588;
        float _15592;
        _15592 = _15589;
        *_15591 = _15592;
        return ;
}

}

#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_5487;
typedef struct {
    struct_Buffer_5487 e0;
    struct_Buffer_5487 e1;
    int e2;
    int e3;
} struct_image_5486;
typedef struct {
    struct_Buffer_5487 e0;
    int e1;
    int e2;
} struct_filter_5490;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_21017(struct_image_5486, struct_filter_5490, struct_Buffer_5487);
__global__ void lambda_21149(struct_filter_5490, struct_image_5486, double*, struct_Buffer_5487, struct_Buffer_5487);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_21017(struct_image_5486 _21020_23356, struct_filter_5490 _21021_23357, struct_Buffer_5487 _21022_23358) {
    __shared__ double ds_img[134][7];
    int  _23364;
    int p_23364;
    int  _23370;
    int p_23370;
    int  _23376;
    int p_23376;
    int  _23382;
    int p_23382;
    int  _23388;
    int p_23388;
    int  _23394;
    int p_23394;
    int  _23417;
    int p_23417;
    double  sum_23419;
    double psum_23419;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _23364 = blockIdx_x();
    p_23364 = _23364;
    l23362: ;
        _23364 = p_23364;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23370 = blockDim_x();
        p_23370 = _23370;
    l23368: ;
        _23370 = p_23370;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23376 = threadIdx_x();
        p_23376 = _23376;
    l23374: ;
        _23376 = p_23376;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23382 = blockIdx_y();
        p_23382 = _23382;
    l23380: ;
        _23382 = p_23382;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23388 = blockDim_y();
        p_23388 = _23388;
    l23386: ;
        _23388 = p_23388;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23394 = threadIdx_y();
        p_23394 = _23394;
    l23392: ;
        _23394 = p_23394;
        #line 11 "main.impala"
        int _23398;
        _23398 = _21020_23356.e2;
        #line 170 "gpu_device.impala"
        int _23395;
        _23395 = _23364 * _23370;
        #line 170 "gpu_device.impala"
        int gid_x_23396;
        gid_x_23396 = _23395 + _23376;
        #line 175 "gpu_device.impala"
        bool _23399;
        _23399 = gid_x_23396 < _23398;
        #line 175 "gpu_device.impala"
        if (_23399) goto l23400; else goto l23465;
    l23465: ;
        #line 178 "gpu_device.impala"
        goto l23464;
    l23400: ;
        #line 11 "main.impala"
        int _23404;
        _23404 = _21020_23356.e3;
        #line 172 "gpu_device.impala"
        int _23401;
        _23401 = _23382 * _23388;
        #line 172 "gpu_device.impala"
        int gid_y_23402;
        gid_y_23402 = _23401 + _23394;
        #line 175 "gpu_device.impala"
        bool _23405;
        _23405 = gid_y_23402 < _23404;
        #line 175 "gpu_device.impala"
        if (_23405) goto l23406; else goto l23463;
    l23463: ;
        #line 178 "gpu_device.impala"
        goto l23464;
    l23464: ;
        return ;
    l23406: ;
        #line 65 "gpu_device.impala"
        int _23436;
        _23436 = gid_y_23402 * _23398;
        #line 4 "gaussian.impala"
        int _23408;
        _23408 = _21021_23357.e1;
        #line 60 "gpu_device.impala"
        char* _23447;
        _23447 = _21022_23358.e1;
        #line 65 "gpu_device.impala"
        struct_Buffer_5487 _23433;
        _23433 = _21020_23356.e1;
        #line 65 "gpu_device.impala"
        int _23449;
        _23449 = _23436 + gid_x_23396;
        #line 4 "gaussian.impala"
        int h_anchor_23410;
        h_anchor_23410 = _23408 / 2;
        #line 60 "gpu_device.impala"
        double* _23448;
        union { double* dst; char* src; } u_23448;
        u_23448.src = _23447;
        _23448 = u_23448.dst;
        #line 65 "gpu_device.impala"
        char* _23434;
        _23434 = _23433.e1;
        #line 60 "gpu_device.impala"
        double* _23450;
        _23450 = _23448 + _23449;
        #line 17 "gaussian.impala"
        bool _23411;
        _23411 = h_anchor_23410 <= gid_x_23396;
        #line 65 "gpu_device.impala"
        double* _23435;
        union { double* dst; char* src; } u_23435;
        u_23435.src = _23434;
        _23435 = u_23435.dst;
        #line 100 "shared_memory_copy"
        for(int i = 0; i < blockDim.x + 6; i += blockDim.x) {
            for(int j = 0; j < blockDim.y + 6; j += blockDim.y) {
                if(threadIdx.x + i < blockDim.x + 6 && 
                   threadIdx.y + j < blockDim.y + 6 && 
                   ((blockIdx.x * blockDim.x + threadIdx.x) - 3 + i) < _21020_23356.e2 && 
                   ((blockIdx.y * blockDim.y + threadIdx.y) - 3 + j) < _21020_23356.e3) {
                    ds_img[threadIdx.x + i][threadIdx.y + j] = \
                      _23435[((blockIdx.y * blockDim.y + threadIdx.y) - 3 + j) * _21020_23356.e2 + ((blockIdx.x * blockDim.x + threadIdx.x) - 3 + i)];
                }
            }
        }
        
        __syncthreads();
        
        #line 17 "gaussian.impala"
        if (_23411) goto l23412; else goto l23462;
    l23462: ;
        #line 27 "gaussian.impala"
        goto l23456;
    l23412: ;
        #line 17 "gaussian.impala"
        int _23413;
        _23413 = _23398 - h_anchor_23410;
        #line 17 "gaussian.impala"
        bool _23414;
        _23414 = gid_x_23396 < _23413;
        #line 17 "gaussian.impala"
        if (_23414) goto l23415; else goto l23455;
    l23455: ;
        #line 27 "gaussian.impala"
        goto l23456;
    l23456: ;
        #line 65 "gpu_device.impala"
        #line 100 "shared_memory_access"
        double* _23457;
        _23457 = &ds_img[_23449 % _21020_23356.e2 + 3 - blockIdx.x * blockDim.x][_23449 / _21020_23356.e2 + 3 - blockIdx.y * blockDim.y];
        #line 65 "gpu_device.impala"
        double _23458;
        _23458 = *_23457;
        #line 65 "gpu_device.impala"
        double _23460;
        _23460 = _23458;
        #line 60 "gpu_device.impala"
        *_23450 = _23460;
        return ;
    l23415: ;
        #line 19 "gaussian.impala"
        int _23421;
        _23421 = 1 + h_anchor_23410;
        #line 19 "gaussian.impala"
        int _23453;
        _23453 = 0 - h_anchor_23410;
        #line 70 "gpu_device.impala"
        struct_Buffer_5487 _23426;
        _23426 = _21021_23357.e0;
        #line 70 "gpu_device.impala"
        char* _23427;
        _23427 = _23426.e1;
        #line 70 "gpu_device.impala"
        double* _23428;
        union { double* dst; char* src; } u_23428;
        u_23428.src = _23427;
        _23428 = u_23428.dst;
        #line 19 "gpu_device.impala"
        p_23417 = _23453;
        psum_23419 = 0.000000e+00;
        goto l23416;
    l23416: ;
        _23417 = p_23417;
        sum_23419 = psum_23419;
        #line 19 "gpu_device.impala"
        bool _23422;
        _23422 = _23417 < _23421;
        #line 19 "gpu_device.impala"
        if (_23422) goto l23423; else goto l23446;
    l23446: ;
        #line 60 "gpu_device.impala"
        *_23450 = sum_23419;
        return ;
    l23423: ;
        #line 21 "gaussian.impala"
        int _23429;
        _23429 = _23417 + h_anchor_23410;
        #line 21 "gaussian.impala"
        int _23437;
        _23437 = gid_x_23396 + _23417;
        #line 69 "gpu_device.impala"
        double* i_23430;
        i_23430 = _23428 + _23429;
        #line 23 "gpu_device.impala"
        int _23424;
        _23424 = 1 + _23417;
        #line 65 "gpu_device.impala"
        int _23438;
        _23438 = _23436 + _23437;
        #line 70 "gpu_device.impala"
        double _23431;
        _23431 = *i_23430;
        #line 65 "gpu_device.impala"
        #line 100 "shared_memory_access"
        double* _23439;
        _23439 = &ds_img[_23438 % _21020_23356.e2 + 3 - blockIdx.x * blockDim.x][_23438 / _21020_23356.e2 + 3 - blockIdx.y * blockDim.y];
        #line 70 "gpu_device.impala"
        double _23442;
        _23442 = _23431;
        #line 65 "gpu_device.impala"
        double _23440;
        _23440 = *_23439;
        #line 65 "gpu_device.impala"
        double _23443;
        _23443 = _23440;
        #line 21 "gaussian.impala"
        double _23444;
        _23444 = _23442 * _23443;
        #line 21 "gaussian.impala"
        double _23445;
        _23445 = sum_23419 + _23444;
        #line 19 "gpu_device.impala"
        p_23417 = _23424;
        psum_23419 = _23445;
        goto l23416;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_21149(struct_filter_5490 _21152_23469, struct_image_5486 _21153_23470, double* _21154_23471, struct_Buffer_5487 _21155_23472, struct_Buffer_5487 _21156_23473) {
    __shared__ double ds_img[134][7];
    int  _23476;
    int p_23476;
    int  _23479;
    int p_23479;
    int  _23482;
    int p_23482;
    int  _23485;
    int p_23485;
    int  _23488;
    int p_23488;
    int  _23491;
    int p_23491;
    int  _23510;
    int p_23510;
    double  sum_23512;
    double psum_23512;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _23476 = blockIdx_x();
    p_23476 = _23476;
    l23474: ;
        _23476 = p_23476;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23479 = blockDim_x();
        p_23479 = _23479;
    l23477: ;
        _23479 = p_23479;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23482 = threadIdx_x();
        p_23482 = _23482;
    l23480: ;
        _23482 = p_23482;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23485 = blockIdx_y();
        p_23485 = _23485;
    l23483: ;
        _23485 = p_23485;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23488 = blockDim_y();
        p_23488 = _23488;
    l23486: ;
        _23488 = p_23488;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23491 = threadIdx_y();
        p_23491 = _23491;
    l23489: ;
        _23491 = p_23491;
        #line 170 "gpu_device.impala"
        int _23492;
        _23492 = _23476 * _23479;
        #line 170 "gpu_device.impala"
        int gid_x_23493;
        gid_x_23493 = _23492 + _23482;
        #line 11 "main.impala"
        int _23494;
        _23494 = _21153_23470.e2;
        #line 175 "gpu_device.impala"
        bool _23495;
        _23495 = gid_x_23493 < _23494;
        #line 175 "gpu_device.impala"
        if (_23495) goto l23496; else goto l23554;
    l23554: ;
        #line 178 "gpu_device.impala"
        goto l23553;
    l23496: ;
        #line 172 "gpu_device.impala"
        int _23497;
        _23497 = _23485 * _23488;
        #line 172 "gpu_device.impala"
        int gid_y_23498;
        gid_y_23498 = _23497 + _23491;
        #line 11 "main.impala"
        int _23499;
        _23499 = _21153_23470.e3;
        #line 175 "gpu_device.impala"
        bool _23500;
        _23500 = gid_y_23498 < _23499;
        #line 175 "gpu_device.impala"
        if (_23500) goto l23501; else goto l23552;
    l23552: ;
        #line 178 "gpu_device.impala"
        goto l23553;
    l23553: ;
        return ;
    l23501: ;
        #line 65 "gpu_device.impala"
        int _23539;
        _23539 = gid_y_23498 * _23494;
        #line 65 "gpu_device.impala"
        int _23540;
        _23540 = _23539 + gid_x_23493;
        #line 6 "gaussian.impala"
        int _23502;
        _23502 = _21152_23469.e2;
        #line 65 "gpu_device.impala"
        char* _23524;
        _23524 = _21156_23473.e1;
        #line 60 "gpu_device.impala"
        char* _23537;
        _23537 = _21155_23472.e1;
        #line 65 "gpu_device.impala"
        double* _23525;
        union { double* dst; char* src; } u_23525;
        u_23525.src = _23524;
        _23525 = u_23525.dst;
        #line 100 "shared_memory_copy"
        for(int i = 0; i < blockDim.x + 6; i += blockDim.x) {
            for(int j = 0; j < blockDim.y + 6; j += blockDim.y) {
                if(threadIdx.x + i < blockDim.x + 6 && 
                   threadIdx.y + j < blockDim.y + 6 && 
                   ((blockIdx.x * blockDim.x + threadIdx.x) - 3 + i) < _21153_23470.e2 && 
                   ((blockIdx.y * blockDim.y + threadIdx.y) - 3 + j) < _21153_23470.e3) {
                    ds_img[threadIdx.x + i][threadIdx.y + j] = \
                      _23525[((blockIdx.y * blockDim.y + threadIdx.y) - 3 + j) * _21153_23470.e2 + ((blockIdx.x * blockDim.x + threadIdx.x) - 3 + i)];
                }
            }
        }
        
        __syncthreads();
        
        #line 6 "gaussian.impala"
        int v_anchor_23503;
        v_anchor_23503 = _23502 / 2;
        #line 60 "gpu_device.impala"
        double* _23538;
        union { double* dst; char* src; } u_23538;
        u_23538.src = _23537;
        _23538 = u_23538.dst;
        #line 39 "gaussian.impala"
        bool _23504;
        _23504 = v_anchor_23503 <= gid_y_23498;
        #line 60 "gpu_device.impala"
        double* _23541;
        _23541 = _23538 + _23540;
        #line 39 "gaussian.impala"
        if (_23504) goto l23505; else goto l23551;
    l23551: ;
        #line 49 "gaussian.impala"
        goto l23545;
    l23505: ;
        #line 39 "gaussian.impala"
        int _23506;
        _23506 = _23499 - v_anchor_23503;
        #line 39 "gaussian.impala"
        bool _23507;
        _23507 = gid_y_23498 < _23506;
        #line 39 "gaussian.impala"
        if (_23507) goto l23508; else goto l23544;
    l23544: ;
        #line 49 "gaussian.impala"
        goto l23545;
    l23545: ;
        #line 65 "gpu_device.impala"
        #line 100 "shared_memory_access"
        double* _23546;
        _23546 = &ds_img[_23540 % _21153_23470.e2 + 3 - blockIdx.x * blockDim.x][_23540 / _21153_23470.e2 + 3 - blockIdx.y * blockDim.y];
        #line 65 "gpu_device.impala"
        double _23547;
        _23547 = *_23546;
        #line 65 "gpu_device.impala"
        double _23549;
        _23549 = _23547;
        #line 60 "gpu_device.impala"
        *_23541 = _23549;
        return ;
    l23508: ;
        #line 41 "gaussian.impala"
        int _23513;
        _23513 = 1 + v_anchor_23503;
        #line 41 "gaussian.impala"
        int _23543;
        _23543 = 0 - v_anchor_23503;
        #line 70 "gpu_device.impala"
        struct_Buffer_5487 _23517;
        _23517 = _21152_23469.e0;
        #line 70 "gpu_device.impala"
        char* _23518;
        _23518 = _23517.e1;
        #line 70 "gpu_device.impala"
        double* _23519;
        union { double* dst; char* src; } u_23519;
        u_23519.src = _23518;
        _23519 = u_23519.dst;
        #line 19 "gpu_device.impala"
        p_23510 = _23543;
        psum_23512 = 0.000000e+00;
        goto l23509;
    l23509: ;
        _23510 = p_23510;
        sum_23512 = psum_23512;
        #line 19 "gpu_device.impala"
        bool _23514;
        _23514 = _23510 < _23513;
        #line 19 "gpu_device.impala"
        if (_23514) goto l23515; else goto l23536;
    l23536: ;
        #line 60 "gpu_device.impala"
        *_23541 = sum_23512;
        return ;
    l23515: ;
        #line 43 "gaussian.impala"
        int _23520;
        _23520 = _23510 + v_anchor_23503;
        #line 69 "gpu_device.impala"
        double* i_23521;
        i_23521 = _23519 + _23520;
        #line 43 "gaussian.impala"
        int _23526;
        _23526 = gid_y_23498 + _23510;
        #line 23 "gpu_device.impala"
        int _23516;
        _23516 = 1 + _23510;
        #line 70 "gpu_device.impala"
        double _23522;
        _23522 = *i_23521;
        #line 65 "gpu_device.impala"
        int _23527;
        _23527 = _23526 * _23494;
        #line 70 "gpu_device.impala"
        double _23532;
        _23532 = _23522;
        #line 65 "gpu_device.impala"
        int _23528;
        _23528 = _23527 + gid_x_23493;
        #line 65 "gpu_device.impala"
        #line 100 "shared_memory_access"
        double* _23529;
        _23529 = &ds_img[_23528 % _21153_23470.e2 + 3 - blockIdx.x * blockDim.x][_23528 / _21153_23470.e2 + 3 - blockIdx.y * blockDim.y];
        #line 65 "gpu_device.impala"
        double _23530;
        _23530 = *_23529;
        #line 65 "gpu_device.impala"
        double _23533;
        _23533 = _23530;
        #line 43 "gaussian.impala"
        double _23534;
        _23534 = _23532 * _23533;
        #line 43 "gaussian.impala"
        double _23535;
        _23535 = sum_23512 + _23534;
        #line 19 "gpu_device.impala"
        p_23510 = _23516;
        psum_23512 = _23535;
        goto l23509;
}

}
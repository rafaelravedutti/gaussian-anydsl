
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_5491;
typedef struct {
    struct_Buffer_5491 e0;
    int e1;
    int e2;
} struct_filter_5490;
typedef struct {
    struct_Buffer_5491 e0;
    struct_Buffer_5491 e1;
    int e2;
    int e3;
} struct_image_5494;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_21041(struct_filter_5490, struct_Buffer_5491, struct_image_5494);
__global__ void lambda_21173(struct_filter_5490, struct_Buffer_5491, double*, struct_Buffer_5491, struct_image_5494);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_21041(struct_filter_5490 _21044_23380, struct_Buffer_5491 _21045_23381, struct_image_5494 _21046_23382) {
    __shared__ double ds_img[134][7];
    __shared__ double ds_filter[7];
    int  _23388;
    int p_23388;
    int  _23394;
    int p_23394;
    int  _23400;
    int p_23400;
    int  _23406;
    int p_23406;
    int  _23412;
    int p_23412;
    int  _23418;
    int p_23418;
    int  _23441;
    int p_23441;
    double  sum_23443;
    double psum_23443;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _23388 = blockIdx_x();
    p_23388 = _23388;
    l23386: ;
        _23388 = p_23388;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23394 = blockDim_x();
        p_23394 = _23394;
    l23392: ;
        _23394 = p_23394;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23400 = threadIdx_x();
        p_23400 = _23400;
    l23398: ;
        _23400 = p_23400;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23406 = blockIdx_y();
        p_23406 = _23406;
    l23404: ;
        _23406 = p_23406;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23412 = blockDim_y();
        p_23412 = _23412;
    l23410: ;
        _23412 = p_23412;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23418 = threadIdx_y();
        p_23418 = _23418;
    l23416: ;
        _23418 = p_23418;
        #line 170 "gpu_device.impala"
        int _23419;
        _23419 = _23388 * _23394;
        #line 11 "main.impala"
        int _23422;
        _23422 = _21046_23382.e2;
        #line 170 "gpu_device.impala"
        int gid_x_23420;
        gid_x_23420 = _23419 + _23400;
        #line 175 "gpu_device.impala"
        bool _23423;
        _23423 = gid_x_23420 < _23422;
        #line 175 "gpu_device.impala"
        if (_23423) goto l23424; else goto l23489;
    l23489: ;
        #line 178 "gpu_device.impala"
        goto l23488;
    l23424: ;
        #line 172 "gpu_device.impala"
        int _23425;
        _23425 = _23406 * _23412;
        #line 172 "gpu_device.impala"
        int gid_y_23426;
        gid_y_23426 = _23425 + _23418;
        #line 11 "main.impala"
        int _23428;
        _23428 = _21046_23382.e3;
        #line 175 "gpu_device.impala"
        bool _23429;
        _23429 = gid_y_23426 < _23428;
        #line 175 "gpu_device.impala"
        if (_23429) goto l23430; else goto l23487;
    l23487: ;
        #line 178 "gpu_device.impala"
        goto l23488;
    l23488: ;
        return ;
    l23430: ;
        #line 65 "gpu_device.impala"
        struct_Buffer_5491 _23457;
        _23457 = _21046_23382.e1;
        #line 60 "gpu_device.impala"
        char* _23471;
        _23471 = _21045_23381.e1;
        #line 65 "gpu_device.impala"
        int _23460;
        _23460 = gid_y_23426 * _23422;
        #line 4 "gaussian.impala"
        int _23432;
        _23432 = _21044_23380.e1;
        #line 4 "gaussian.impala"
        int h_anchor_23434;
        h_anchor_23434 = _23432 / 2;
        #line 65 "gpu_device.impala"
        int _23473;
        _23473 = _23460 + gid_x_23420;
        #line 65 "gpu_device.impala"
        char* _23458;
        _23458 = _23457.e1;
        #line 60 "gpu_device.impala"
        double* _23472;
        union { double* dst; char* src; } u_23472;
        u_23472.src = _23471;
        _23472 = u_23472.dst;
        #line 17 "gaussian.impala"
        bool _23435;
        _23435 = h_anchor_23434 <= gid_x_23420;
        #line 60 "gpu_device.impala"
        double* _23474;
        _23474 = _23472 + _23473;
        #line 65 "gpu_device.impala"
        double* _23459;
        union { double* dst; char* src; } u_23459;
        u_23459.src = _23458;
        _23459 = u_23459.dst;
        #line 100 "shared_memory_image_copy"
        for(int i = 0; i < blockDim.x + 6; i += blockDim.x) {
            for(int j = 0; j < blockDim.y + 6; j += blockDim.y) {
                if(threadIdx.x + i < blockDim.x + 6 && 
                   threadIdx.y + j < blockDim.y + 6 && 
                   ((blockIdx.x * blockDim.x + threadIdx.x) - 3 + i) < _21046_23382.e2 && 
                   ((blockIdx.y * blockDim.y + threadIdx.y) - 3 + j) < _21046_23382.e3) {
                    ds_img[threadIdx.x + i][threadIdx.y + j] = \
                      _23459[((blockIdx.y * blockDim.y + threadIdx.y) - 3 + j) * _21046_23382.e2 + ((blockIdx.x * blockDim.x + threadIdx.x) - 3 + i)];
                }
            }
        }
        
        __syncthreads();
        
        #line 17 "gaussian.impala"
        if (_23435) goto l23436; else goto l23486;
    l23486: ;
        #line 27 "gaussian.impala"
        goto l23480;
    l23436: ;
        #line 17 "gaussian.impala"
        int _23437;
        _23437 = _23422 - h_anchor_23434;
        #line 17 "gaussian.impala"
        bool _23438;
        _23438 = gid_x_23420 < _23437;
        #line 17 "gaussian.impala"
        if (_23438) goto l23439; else goto l23479;
    l23479: ;
        #line 27 "gaussian.impala"
        goto l23480;
    l23480: ;
        #line 65 "gpu_device.impala"
        #line 100 "shared_memory_access"
        double* _23481;
        _23481 = &ds_img[_23473 % _21046_23382.e2 + 3 - blockIdx.x * blockDim.x][_23473 / _21046_23382.e2 + 3 - blockIdx.y * blockDim.y];
        #line 65 "gpu_device.impala"
        double _23482;
        _23482 = *_23481;
        #line 65 "gpu_device.impala"
        double _23484;
        _23484 = _23482;
        #line 60 "gpu_device.impala"
        *_23474 = _23484;
        return ;
    l23439: ;
        #line 70 "gpu_device.impala"
        struct_Buffer_5491 _23450;
        _23450 = _21044_23380.e0;
        #line 19 "gaussian.impala"
        int _23477;
        _23477 = 0 - h_anchor_23434;
        #line 19 "gaussian.impala"
        int _23445;
        _23445 = 1 + h_anchor_23434;
        #line 70 "gpu_device.impala"
        char* _23451;
        _23451 = _23450.e1;
        #line 70 "gpu_device.impala"
        double* _23452;
        union { double* dst; char* src; } u_23452;
        u_23452.src = _23451;
        _23452 = u_23452.dst;
        #line 200 "shared_memory_filter_copy"
        for(int i = 0; i < 7; i++) {
            ds_filter[i] = _23452[i];
        }
        
        __syncthreads();
        
        #line 19 "gpu_device.impala"
        p_23441 = _23477;
        psum_23443 = 0.000000e+00;
        goto l23440;
    l23440: ;
        _23441 = p_23441;
        sum_23443 = psum_23443;
        #line 19 "gpu_device.impala"
        bool _23446;
        _23446 = _23441 < _23445;
        #line 19 "gpu_device.impala"
        if (_23446) goto l23447; else goto l23470;
    l23470: ;
        #line 60 "gpu_device.impala"
        *_23474 = sum_23443;
        return ;
    l23447: ;
        #line 21 "gaussian.impala"
        int _23461;
        _23461 = gid_x_23420 + _23441;
        #line 21 "gaussian.impala"
        int _23453;
        _23453 = _23441 + h_anchor_23434;
        #line 23 "gpu_device.impala"
        int _23448;
        _23448 = 1 + _23441;
        #line 65 "gpu_device.impala"
        int _23462;
        _23462 = _23460 + _23461;
        #line 69 "gpu_device.impala"
        #line 100 "shared_memory_access"
        double* i_23454;
        i_23454 = &ds_filter[_23453];
        #line 65 "gpu_device.impala"
        #line 100 "shared_memory_access"
        double* _23463;
        _23463 = &ds_img[_23462 % _21046_23382.e2 + 3 - blockIdx.x * blockDim.x][_23462 / _21046_23382.e2 + 3 - blockIdx.y * blockDim.y];
        #line 70 "gpu_device.impala"
        double _23455;
        _23455 = *i_23454;
        #line 70 "gpu_device.impala"
        double _23466;
        _23466 = _23455;
        #line 65 "gpu_device.impala"
        double _23464;
        _23464 = *_23463;
        #line 65 "gpu_device.impala"
        double _23467;
        _23467 = _23464;
        #line 21 "gaussian.impala"
        double _23468;
        _23468 = _23466 * _23467;
        #line 21 "gaussian.impala"
        double _23469;
        _23469 = sum_23443 + _23468;
        #line 19 "gpu_device.impala"
        p_23441 = _23448;
        psum_23443 = _23469;
        goto l23440;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_21173(struct_filter_5490 _21176_23493, struct_Buffer_5491 _21177_23494, double* _21178_23495, struct_Buffer_5491 _21179_23496, struct_image_5494 _21180_23497) {
    __shared__ double ds_img[134][7];
    __shared__ double ds_filter[7];
    int  _23500;
    int p_23500;
    int  _23503;
    int p_23503;
    int  _23506;
    int p_23506;
    int  _23509;
    int p_23509;
    int  _23512;
    int p_23512;
    int  _23515;
    int p_23515;
    int  _23534;
    int p_23534;
    double  sum_23536;
    double psum_23536;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _23500 = blockIdx_x();
    p_23500 = _23500;
    l23498: ;
        _23500 = p_23500;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23503 = blockDim_x();
        p_23503 = _23503;
    l23501: ;
        _23503 = p_23503;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23506 = threadIdx_x();
        p_23506 = _23506;
    l23504: ;
        _23506 = p_23506;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23509 = blockIdx_y();
        p_23509 = _23509;
    l23507: ;
        _23509 = p_23509;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23512 = blockDim_y();
        p_23512 = _23512;
    l23510: ;
        _23512 = p_23512;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23515 = threadIdx_y();
        p_23515 = _23515;
    l23513: ;
        _23515 = p_23515;
        #line 11 "main.impala"
        int _23518;
        _23518 = _21180_23497.e2;
        #line 170 "gpu_device.impala"
        int _23516;
        _23516 = _23500 * _23503;
        #line 170 "gpu_device.impala"
        int gid_x_23517;
        gid_x_23517 = _23516 + _23506;
        #line 175 "gpu_device.impala"
        bool _23519;
        _23519 = gid_x_23517 < _23518;
        #line 175 "gpu_device.impala"
        if (_23519) goto l23520; else goto l23578;
    l23578: ;
        #line 178 "gpu_device.impala"
        goto l23577;
    l23520: ;
        #line 172 "gpu_device.impala"
        int _23521;
        _23521 = _23509 * _23512;
        #line 11 "main.impala"
        int _23523;
        _23523 = _21180_23497.e3;
        #line 172 "gpu_device.impala"
        int gid_y_23522;
        gid_y_23522 = _23521 + _23515;
        #line 175 "gpu_device.impala"
        bool _23524;
        _23524 = gid_y_23522 < _23523;
        #line 175 "gpu_device.impala"
        if (_23524) goto l23525; else goto l23576;
    l23576: ;
        #line 178 "gpu_device.impala"
        goto l23577;
    l23577: ;
        return ;
    l23525: ;
        #line 65 "gpu_device.impala"
        int _23563;
        _23563 = gid_y_23522 * _23518;
        #line 60 "gpu_device.impala"
        char* _23561;
        _23561 = _21177_23494.e1;
        #line 6 "gaussian.impala"
        int _23526;
        _23526 = _21176_23493.e2;
        #line 65 "gpu_device.impala"
        char* _23548;
        _23548 = _21179_23496.e1;
        #line 65 "gpu_device.impala"
        int _23564;
        _23564 = _23563 + gid_x_23517;
        #line 60 "gpu_device.impala"
        double* _23562;
        union { double* dst; char* src; } u_23562;
        u_23562.src = _23561;
        _23562 = u_23562.dst;
        #line 6 "gaussian.impala"
        int v_anchor_23527;
        v_anchor_23527 = _23526 / 2;
        #line 65 "gpu_device.impala"
        double* _23549;
        union { double* dst; char* src; } u_23549;
        u_23549.src = _23548;
        _23549 = u_23549.dst;
        #line 100 "shared_memory_image_copy"
        for(int i = 0; i < blockDim.x + 6; i += blockDim.x) {
            for(int j = 0; j < blockDim.y + 6; j += blockDim.y) {
                if(threadIdx.x + i < blockDim.x + 6 && 
                   threadIdx.y + j < blockDim.y + 6 && 
                   ((blockIdx.x * blockDim.x + threadIdx.x) - 3 + i) < _21180_23497.e2 && 
                   ((blockIdx.y * blockDim.y + threadIdx.y) - 3 + j) < _21180_23497.e3) {
                    ds_img[threadIdx.x + i][threadIdx.y + j] = \
                      _23549[((blockIdx.y * blockDim.y + threadIdx.y) - 3 + j) * _21180_23497.e2 + ((blockIdx.x * blockDim.x + threadIdx.x) - 3 + i)];
                }
            }
        }
        
        __syncthreads();
        
        #line 60 "gpu_device.impala"
        double* _23565;
        _23565 = _23562 + _23564;
        #line 39 "gaussian.impala"
        bool _23528;
        _23528 = v_anchor_23527 <= gid_y_23522;
        #line 39 "gaussian.impala"
        if (_23528) goto l23529; else goto l23575;
    l23575: ;
        #line 49 "gaussian.impala"
        goto l23569;
    l23529: ;
        #line 39 "gaussian.impala"
        int _23530;
        _23530 = _23523 - v_anchor_23527;
        #line 39 "gaussian.impala"
        bool _23531;
        _23531 = gid_y_23522 < _23530;
        #line 39 "gaussian.impala"
        if (_23531) goto l23532; else goto l23568;
    l23568: ;
        #line 49 "gaussian.impala"
        goto l23569;
    l23569: ;
        #line 65 "gpu_device.impala"
        #line 100 "shared_memory_access"
        double* _23570;
        _23570 = &ds_img[_23564 % _21180_23497.e2 + 3 - blockIdx.x * blockDim.x][_23564 / _21180_23497.e2 + 3 - blockIdx.y * blockDim.y];
        #line 65 "gpu_device.impala"
        double _23571;
        _23571 = *_23570;
        #line 65 "gpu_device.impala"
        double _23573;
        _23573 = _23571;
        #line 60 "gpu_device.impala"
        *_23565 = _23573;
        return ;
    l23532: ;
        #line 41 "gaussian.impala"
        int _23567;
        _23567 = 0 - v_anchor_23527;
        #line 70 "gpu_device.impala"
        struct_Buffer_5491 _23541;
        _23541 = _21176_23493.e0;
        #line 41 "gaussian.impala"
        int _23537;
        _23537 = 1 + v_anchor_23527;
        #line 70 "gpu_device.impala"
        char* _23542;
        _23542 = _23541.e1;
        #line 70 "gpu_device.impala"
        double* _23543;
        union { double* dst; char* src; } u_23543;
        u_23543.src = _23542;
        _23543 = u_23543.dst;
        #line 200 "shared_memory_filter_copy"
        for(int i = 0; i < 7; i++) {
            ds_filter[i] = _23543[i];
        }
        
        __syncthreads();
        
        #line 19 "gpu_device.impala"
        p_23534 = _23567;
        psum_23536 = 0.000000e+00;
        goto l23533;
    l23533: ;
        _23534 = p_23534;
        sum_23536 = psum_23536;
        #line 19 "gpu_device.impala"
        bool _23538;
        _23538 = _23534 < _23537;
        #line 19 "gpu_device.impala"
        if (_23538) goto l23539; else goto l23560;
    l23560: ;
        #line 60 "gpu_device.impala"
        *_23565 = sum_23536;
        return ;
    l23539: ;
        #line 23 "gpu_device.impala"
        int _23540;
        _23540 = 1 + _23534;
        #line 43 "gaussian.impala"
        int _23550;
        _23550 = gid_y_23522 + _23534;
        #line 43 "gaussian.impala"
        int _23544;
        _23544 = _23534 + v_anchor_23527;
        #line 65 "gpu_device.impala"
        int _23551;
        _23551 = _23550 * _23518;
        #line 69 "gpu_device.impala"
        #line 100 "shared_memory_access"
        double* i_23545;
        i_23545 = &ds_filter[_23544];
        #line 65 "gpu_device.impala"
        int _23552;
        _23552 = _23551 + gid_x_23517;
        #line 70 "gpu_device.impala"
        double _23546;
        _23546 = *i_23545;
        #line 65 "gpu_device.impala"
        #line 100 "shared_memory_access"
        double* _23553;
        _23553 = &ds_img[_23552 % _21180_23497.e2 + 3 - blockIdx.x * blockDim.x][_23552 / _21180_23497.e2 + 3 - blockIdx.y * blockDim.y];
        #line 70 "gpu_device.impala"
        double _23556;
        _23556 = _23546;
        #line 65 "gpu_device.impala"
        double _23554;
        _23554 = *_23553;
        #line 65 "gpu_device.impala"
        double _23557;
        _23557 = _23554;
        #line 43 "gaussian.impala"
        double _23558;
        _23558 = _23556 * _23557;
        #line 43 "gaussian.impala"
        double _23559;
        _23559 = sum_23536 + _23558;
        #line 19 "gpu_device.impala"
        p_23534 = _23540;
        psum_23536 = _23559;
        goto l23533;
}

}
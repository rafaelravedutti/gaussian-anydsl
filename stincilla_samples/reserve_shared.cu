
#include <hip/hip_runtime.h>
extern "C" {
__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_11973(float*, float*);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_11973(float* _11976_13141, float* _11977_13142) {
    float*  shared_13150;
    float* pshared_13150;
    int  threadIdx_x_13156;
    int pthreadIdx_x_13156;
    int  blockDim_x_13162;
    int pblockDim_x_13162;
    int  blockIdx_x_13168;
    int pblockIdx_x_13168;
    int  threadIdx_y_13174;
    int pthreadIdx_y_13174;
    int  blockDim_y_13180;
    int pblockDim_y_13180;
    int  blockIdx_y_13186;
    int pblockIdx_y_13186;
    int  _13200;
    int p_13200;
    int  threadIdx_x_13207;
    int pthreadIdx_x_13207;
    int  blockDim_x_13210;
    int pblockDim_x_13210;
    int  blockIdx_x_13213;
    int pblockIdx_x_13213;
    int  threadIdx_y_13216;
    int pthreadIdx_y_13216;
    int  blockDim_y_13219;
    int pblockDim_y_13219;
    int  blockIdx_y_13222;
    int pblockIdx_y_13222;
    int  _13225;
    int p_13225;
    __shared__ float reserver_shared_13150[128];
    pshared_13150 = reserver_shared_13150;
    l13148: ;
        shared_13150 = pshared_13150;
        threadIdx_x_13156 = threadIdx_x();
        pthreadIdx_x_13156 = threadIdx_x_13156;
    l13154: ;
        threadIdx_x_13156 = pthreadIdx_x_13156;
        blockDim_x_13162 = blockDim_x();
        pblockDim_x_13162 = blockDim_x_13162;
    l13160: ;
        blockDim_x_13162 = pblockDim_x_13162;
        blockIdx_x_13168 = blockIdx_x();
        pblockIdx_x_13168 = blockIdx_x_13168;
    l13166: ;
        blockIdx_x_13168 = pblockIdx_x_13168;
        threadIdx_y_13174 = threadIdx_y();
        pthreadIdx_y_13174 = threadIdx_y_13174;
    l13172: ;
        threadIdx_y_13174 = pthreadIdx_y_13174;
        blockDim_y_13180 = blockDim_y();
        pblockDim_y_13180 = blockDim_y_13180;
    l13178: ;
        blockDim_y_13180 = pblockDim_y_13180;
        blockIdx_y_13186 = blockIdx_y();
        pblockIdx_y_13186 = blockIdx_y_13186;
    l13184: ;
        blockIdx_y_13186 = pblockIdx_y_13186;
        int _13188;
        _13188 = blockDim_y_13180 * blockIdx_y_13186;
        int _13191;
        _13191 = blockDim_x_13162 * blockIdx_x_13168;
        int _13189;
        _13189 = threadIdx_y_13174 + _13188;
        int _13192;
        _13192 = threadIdx_x_13156 + _13191;
        int _13190;
        _13190 = 2048 * _13189;
        int _13193;
        _13193 = _13190 + _13192;
        float* _13194;
        _13194 = _11976_13141 + _13193;
        float _13195;
        _13195 = *_13194;
        _13200 = threadIdx_x();
        p_13200 = _13200;
    l13198: ;
        _13200 = p_13200;
        float* _13201;
        _13201 = shared_13150 + _13200;
        float _13203;
        _13203 = _13195;
        *_13201 = _13203;
        threadIdx_x_13207 = threadIdx_x();
        pthreadIdx_x_13207 = threadIdx_x_13207;
    l13205: ;
        threadIdx_x_13207 = pthreadIdx_x_13207;
        blockDim_x_13210 = blockDim_x();
        pblockDim_x_13210 = blockDim_x_13210;
    l13208: ;
        blockDim_x_13210 = pblockDim_x_13210;
        blockIdx_x_13213 = blockIdx_x();
        pblockIdx_x_13213 = blockIdx_x_13213;
    l13211: ;
        blockIdx_x_13213 = pblockIdx_x_13213;
        threadIdx_y_13216 = threadIdx_y();
        pthreadIdx_y_13216 = threadIdx_y_13216;
    l13214: ;
        threadIdx_y_13216 = pthreadIdx_y_13216;
        blockDim_y_13219 = blockDim_y();
        pblockDim_y_13219 = blockDim_y_13219;
    l13217: ;
        blockDim_y_13219 = pblockDim_y_13219;
        blockIdx_y_13222 = blockIdx_y();
        pblockIdx_y_13222 = blockIdx_y_13222;
    l13220: ;
        blockIdx_y_13222 = pblockIdx_y_13222;
        _13225 = threadIdx_x();
        p_13225 = _13225;
    l13223: ;
        _13225 = p_13225;
        int _13229;
        _13229 = blockDim_y_13219 * blockIdx_y_13222;
        float* _13226;
        _13226 = shared_13150 + _13225;
        int _13230;
        _13230 = threadIdx_y_13216 + _13229;
        int _13232;
        _13232 = blockDim_x_13210 * blockIdx_x_13213;
        float _13227;
        _13227 = *_13226;
        int _13231;
        _13231 = 2048 * _13230;
        int _13233;
        _13233 = threadIdx_x_13207 + _13232;
        float _13236;
        _13236 = _13227;
        int _13234;
        _13234 = _13231 + _13233;
        float* _13235;
        _13235 = _11977_13142 + _13234;
        *_13235 = _13236;
        return ;
}

}
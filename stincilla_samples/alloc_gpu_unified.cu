
#include <hip/hip_runtime.h>
extern "C" {
__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_9740(float*, float*);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_9740(float* _9743_10463, float* _9744_10464) {
    int  threadIdx_x_10470;
    int pthreadIdx_x_10470;
    int  blockDim_x_10476;
    int pblockDim_x_10476;
    int  blockIdx_x_10482;
    int pblockIdx_x_10482;
    int  threadIdx_y_10488;
    int pthreadIdx_y_10488;
    int  blockDim_y_10494;
    int pblockDim_y_10494;
    int  blockIdx_y_10500;
    int pblockIdx_y_10500;
    threadIdx_x_10470 = threadIdx_x();
    pthreadIdx_x_10470 = threadIdx_x_10470;
    l10468: ;
        threadIdx_x_10470 = pthreadIdx_x_10470;
        blockDim_x_10476 = blockDim_x();
        pblockDim_x_10476 = blockDim_x_10476;
    l10474: ;
        blockDim_x_10476 = pblockDim_x_10476;
        blockIdx_x_10482 = blockIdx_x();
        pblockIdx_x_10482 = blockIdx_x_10482;
    l10480: ;
        blockIdx_x_10482 = pblockIdx_x_10482;
        threadIdx_y_10488 = threadIdx_y();
        pthreadIdx_y_10488 = threadIdx_y_10488;
    l10486: ;
        threadIdx_y_10488 = pthreadIdx_y_10488;
        blockDim_y_10494 = blockDim_y();
        pblockDim_y_10494 = blockDim_y_10494;
    l10492: ;
        blockDim_y_10494 = pblockDim_y_10494;
        blockIdx_y_10500 = blockIdx_y();
        pblockIdx_y_10500 = blockIdx_y_10500;
    l10498: ;
        blockIdx_y_10500 = pblockIdx_y_10500;
        int _10502;
        _10502 = blockDim_y_10494 * blockIdx_y_10500;
        int _10505;
        _10505 = blockDim_x_10476 * blockIdx_x_10482;
        int _10503;
        _10503 = threadIdx_y_10488 + _10502;
        int _10506;
        _10506 = threadIdx_x_10470 + _10505;
        int _10504;
        _10504 = 2048 * _10503;
        int idx_10507;
        idx_10507 = _10504 + _10506;
        float* i_10512;
        i_10512 = _9744_10464 + idx_10507;
        float* i_10508;
        i_10508 = _9743_10463 + idx_10507;
        float _10509;
        _10509 = *i_10508;
        float _10514;
        _10514 = _10509;
        *i_10512 = _10514;
        return ;
}

}
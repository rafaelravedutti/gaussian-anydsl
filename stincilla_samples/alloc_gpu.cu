
#include <hip/hip_runtime.h>
extern "C" {
__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_10603(float*, float*);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_10603(float* _10606_11332, float* _10607_11333) {
    int  threadIdx_x_11339;
    int pthreadIdx_x_11339;
    int  blockDim_x_11345;
    int pblockDim_x_11345;
    int  blockIdx_x_11351;
    int pblockIdx_x_11351;
    int  threadIdx_y_11357;
    int pthreadIdx_y_11357;
    int  blockDim_y_11363;
    int pblockDim_y_11363;
    int  blockIdx_y_11369;
    int pblockIdx_y_11369;
    threadIdx_x_11339 = threadIdx_x();
    pthreadIdx_x_11339 = threadIdx_x_11339;
    l11337: ;
        threadIdx_x_11339 = pthreadIdx_x_11339;
        blockDim_x_11345 = blockDim_x();
        pblockDim_x_11345 = blockDim_x_11345;
    l11343: ;
        blockDim_x_11345 = pblockDim_x_11345;
        blockIdx_x_11351 = blockIdx_x();
        pblockIdx_x_11351 = blockIdx_x_11351;
    l11349: ;
        blockIdx_x_11351 = pblockIdx_x_11351;
        threadIdx_y_11357 = threadIdx_y();
        pthreadIdx_y_11357 = threadIdx_y_11357;
    l11355: ;
        threadIdx_y_11357 = pthreadIdx_y_11357;
        blockDim_y_11363 = blockDim_y();
        pblockDim_y_11363 = blockDim_y_11363;
    l11361: ;
        blockDim_y_11363 = pblockDim_y_11363;
        blockIdx_y_11369 = blockIdx_y();
        pblockIdx_y_11369 = blockIdx_y_11369;
    l11367: ;
        blockIdx_y_11369 = pblockIdx_y_11369;
        int _11371;
        _11371 = blockDim_y_11363 * blockIdx_y_11369;
        int _11374;
        _11374 = blockDim_x_11345 * blockIdx_x_11351;
        int _11372;
        _11372 = threadIdx_y_11357 + _11371;
        int _11375;
        _11375 = threadIdx_x_11339 + _11374;
        int _11373;
        _11373 = 2048 * _11372;
        int idx_11376;
        idx_11376 = _11373 + _11375;
        float* i_11381;
        i_11381 = _10607_11333 + idx_11376;
        float* i_11377;
        i_11377 = _10606_11332 + idx_11376;
        float _11378;
        _11378 = *i_11377;
        float _11383;
        _11383 = _11378;
        *i_11381 = _11383;
        return ;
}

}

#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_5838;
typedef struct {
    struct_Buffer_5838 e0;
    struct_Buffer_5838 e1;
    int e2;
    int e3;
} struct_image_5837;
typedef struct {
    struct_Buffer_5838 e0;
    int e1;
    int e2;
} struct_filter_5841;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_21887();
__global__ void lambda_22080(struct_image_5837, struct_Buffer_5838, struct_filter_5841);
__global__ void lambda_22212(double*, struct_image_5837, struct_Buffer_5838, struct_Buffer_5838, struct_filter_5841);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_21887() {
    return ;
}

__global__ __launch_bounds__ (16 * 16 * 1) void lambda_22080(struct_image_5837 _22083_24475, struct_Buffer_5838 _22084_24476, struct_filter_5841 _22085_24477) {
    int  _24483;
    int p_24483;
    int  _24489;
    int p_24489;
    int  _24495;
    int p_24495;
    int  _24501;
    int p_24501;
    int  _24507;
    int p_24507;
    int  _24513;
    int p_24513;
    int  _24536;
    int p_24536;
    double  sum_24538;
    double psum_24538;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _24483 = blockIdx_x();
    p_24483 = _24483;
    l24481: ;
        _24483 = p_24483;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _24489 = blockDim_x();
        p_24489 = _24489;
    l24487: ;
        _24489 = p_24489;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _24495 = threadIdx_x();
        p_24495 = _24495;
    l24493: ;
        _24495 = p_24495;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _24501 = blockIdx_y();
        p_24501 = _24501;
    l24499: ;
        _24501 = p_24501;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _24507 = blockDim_y();
        p_24507 = _24507;
    l24505: ;
        _24507 = p_24507;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _24513 = threadIdx_y();
        p_24513 = _24513;
    l24511: ;
        _24513 = p_24513;
        #line 185 "impala/gpu_device.impala"
        int _24514;
        _24514 = _24483 * _24489;
        #line 11 "impala/main.impala"
        int _24517;
        _24517 = _22083_24475.e2;
        #line 185 "impala/gpu_device.impala"
        int gid_x_24515;
        gid_x_24515 = _24514 + _24495;
        #line 190 "impala/gpu_device.impala"
        bool _24518;
        _24518 = gid_x_24515 < _24517;
        #line 190 "impala/gpu_device.impala"
        if (_24518) goto l24519; else goto l24584;
    l24584: ;
        #line 193 "impala/gpu_device.impala"
        goto l24583;
    l24519: ;
        #line 187 "impala/gpu_device.impala"
        int _24520;
        _24520 = _24501 * _24507;
        #line 11 "impala/main.impala"
        int _24523;
        _24523 = _22083_24475.e3;
        #line 187 "impala/gpu_device.impala"
        int gid_y_24521;
        gid_y_24521 = _24520 + _24513;
        #line 190 "impala/gpu_device.impala"
        bool _24524;
        _24524 = gid_y_24521 < _24523;
        #line 190 "impala/gpu_device.impala"
        if (_24524) goto l24525; else goto l24582;
    l24582: ;
        #line 193 "impala/gpu_device.impala"
        goto l24583;
    l24583: ;
        return ;
    l24525: ;
        #line 65 "impala/gpu_device.impala"
        struct_Buffer_5838 _24552;
        _24552 = _22083_24475.e1;
        #line 65 "impala/gpu_device.impala"
        int _24555;
        _24555 = gid_y_24521 * _24517;
        #line 9 "impala/gaussian.impala"
        int _24527;
        _24527 = _22085_24477.e1;
        #line 9 "impala/gaussian.impala"
        int h_anchor_24529;
        h_anchor_24529 = _24527 / 2;
        #line 60 "impala/gpu_device.impala"
        char* _24566;
        _24566 = _22084_24476.e1;
        #line 65 "impala/gpu_device.impala"
        char* _24553;
        _24553 = _24552.e1;
        #line 65 "impala/gpu_device.impala"
        int _24568;
        _24568 = _24555 + gid_x_24515;
        #line 22 "impala/gaussian.impala"
        bool _24530;
        _24530 = h_anchor_24529 <= gid_x_24515;
        #line 60 "impala/gpu_device.impala"
        double* _24567;
        union { double* dst; char* src; } u_24567;
        u_24567.src = _24566;
        _24567 = u_24567.dst;
        #line 65 "impala/gpu_device.impala"
        double* _24554;
        union { double* dst; char* src; } u_24554;
        u_24554.src = _24553;
        _24554 = u_24554.dst;
        #line 60 "impala/gpu_device.impala"
        double* _24569;
        _24569 = _24567 + _24568;
        #line 22 "impala/gaussian.impala"
        if (_24530) goto l24531; else goto l24581;
    l24581: ;
        #line 32 "impala/gaussian.impala"
        goto l24575;
    l24531: ;
        #line 22 "impala/gaussian.impala"
        int _24532;
        _24532 = _24517 - h_anchor_24529;
        #line 22 "impala/gaussian.impala"
        bool _24533;
        _24533 = gid_x_24515 < _24532;
        #line 22 "impala/gaussian.impala"
        if (_24533) goto l24534; else goto l24574;
    l24574: ;
        #line 32 "impala/gaussian.impala"
        goto l24575;
    l24575: ;
        #line 65 "impala/gpu_device.impala"
        double* _24576;
        _24576 = _24554 + _24568;
        #line 65 "impala/gpu_device.impala"
        double _24577;
        _24577 = *_24576;
        #line 65 "impala/gpu_device.impala"
        double _24579;
        _24579 = _24577;
        #line 60 "impala/gpu_device.impala"
        *_24569 = _24579;
        return ;
    l24534: ;
        #line 24 "impala/gaussian.impala"
        int _24572;
        _24572 = 0 - h_anchor_24529;
        #line 70 "impala/gpu_device.impala"
        struct_Buffer_5838 _24545;
        _24545 = _22085_24477.e0;
        #line 24 "impala/gaussian.impala"
        int _24540;
        _24540 = 1 + h_anchor_24529;
        #line 70 "impala/gpu_device.impala"
        char* _24546;
        _24546 = _24545.e1;
        #line 70 "impala/gpu_device.impala"
        double* _24547;
        union { double* dst; char* src; } u_24547;
        u_24547.src = _24546;
        _24547 = u_24547.dst;
        #line 19 "impala/gpu_device.impala"
        p_24536 = _24572;
        psum_24538 = 0.000000e+00;
        goto l24535;
    l24535: ;
        _24536 = p_24536;
        sum_24538 = psum_24538;
        #line 19 "impala/gpu_device.impala"
        bool _24541;
        _24541 = _24536 < _24540;
        #line 19 "impala/gpu_device.impala"
        if (_24541) goto l24542; else goto l24565;
    l24565: ;
        #line 60 "impala/gpu_device.impala"
        *_24569 = sum_24538;
        return ;
    l24542: ;
        #line 26 "impala/gaussian.impala"
        int _24556;
        _24556 = gid_x_24515 + _24536;
        #line 23 "impala/gpu_device.impala"
        int _24543;
        _24543 = 1 + _24536;
        #line 65 "impala/gpu_device.impala"
        int _24557;
        _24557 = _24555 + _24556;
        #line 26 "impala/gaussian.impala"
        int _24548;
        _24548 = _24536 + h_anchor_24529;
        #line 69 "impala/gpu_device.impala"
        double* i_24549;
        i_24549 = _24547 + _24548;
        #line 65 "impala/gpu_device.impala"
        double* _24558;
        _24558 = _24554 + _24557;
        #line 70 "impala/gpu_device.impala"
        double _24550;
        _24550 = *i_24549;
        #line 70 "impala/gpu_device.impala"
        double _24561;
        _24561 = _24550;
        #line 65 "impala/gpu_device.impala"
        double _24559;
        _24559 = *_24558;
        #line 65 "impala/gpu_device.impala"
        double _24562;
        _24562 = _24559;
        #line 26 "impala/gaussian.impala"
        double _24563;
        _24563 = _24561 * _24562;
        #line 26 "impala/gaussian.impala"
        double _24564;
        _24564 = sum_24538 + _24563;
        #line 19 "impala/gpu_device.impala"
        p_24536 = _24543;
        psum_24538 = _24564;
        goto l24535;
}

__global__ __launch_bounds__ (16 * 16 * 1) void lambda_22212(double* _22215_24588, struct_image_5837 _22216_24589, struct_Buffer_5838 _22217_24590, struct_Buffer_5838 _22218_24591, struct_filter_5841 _22219_24592) {
    int  _24595;
    int p_24595;
    int  _24598;
    int p_24598;
    int  _24601;
    int p_24601;
    int  _24604;
    int p_24604;
    int  _24607;
    int p_24607;
    int  _24610;
    int p_24610;
    int  _24629;
    int p_24629;
    double  sum_24631;
    double psum_24631;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _24595 = blockIdx_x();
    p_24595 = _24595;
    l24593: ;
        _24595 = p_24595;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _24598 = blockDim_x();
        p_24598 = _24598;
    l24596: ;
        _24598 = p_24598;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _24601 = threadIdx_x();
        p_24601 = _24601;
    l24599: ;
        _24601 = p_24601;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _24604 = blockIdx_y();
        p_24604 = _24604;
    l24602: ;
        _24604 = p_24604;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _24607 = blockDim_y();
        p_24607 = _24607;
    l24605: ;
        _24607 = p_24607;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _24610 = threadIdx_y();
        p_24610 = _24610;
    l24608: ;
        _24610 = p_24610;
        #line 11 "impala/main.impala"
        int _24613;
        _24613 = _22216_24589.e2;
        #line 185 "impala/gpu_device.impala"
        int _24611;
        _24611 = _24595 * _24598;
        #line 185 "impala/gpu_device.impala"
        int gid_x_24612;
        gid_x_24612 = _24611 + _24601;
        #line 190 "impala/gpu_device.impala"
        bool _24614;
        _24614 = gid_x_24612 < _24613;
        #line 190 "impala/gpu_device.impala"
        if (_24614) goto l24615; else goto l24673;
    l24673: ;
        #line 193 "impala/gpu_device.impala"
        goto l24672;
    l24615: ;
        #line 11 "impala/main.impala"
        int _24618;
        _24618 = _22216_24589.e3;
        #line 187 "impala/gpu_device.impala"
        int _24616;
        _24616 = _24604 * _24607;
        #line 187 "impala/gpu_device.impala"
        int gid_y_24617;
        gid_y_24617 = _24616 + _24610;
        #line 190 "impala/gpu_device.impala"
        bool _24619;
        _24619 = gid_y_24617 < _24618;
        #line 190 "impala/gpu_device.impala"
        if (_24619) goto l24620; else goto l24671;
    l24671: ;
        #line 193 "impala/gpu_device.impala"
        goto l24672;
    l24672: ;
        return ;
    l24620: ;
        #line 65 "impala/gpu_device.impala"
        int _24658;
        _24658 = gid_y_24617 * _24613;
        #line 11 "impala/gaussian.impala"
        int _24621;
        _24621 = _22219_24592.e2;
        #line 60 "impala/gpu_device.impala"
        char* _24656;
        _24656 = _22218_24591.e1;
        #line 60 "impala/gpu_device.impala"
        double* _24657;
        union { double* dst; char* src; } u_24657;
        u_24657.src = _24656;
        _24657 = u_24657.dst;
        #line 65 "impala/gpu_device.impala"
        char* _24643;
        _24643 = _22217_24590.e1;
        #line 65 "impala/gpu_device.impala"
        int _24659;
        _24659 = _24658 + gid_x_24612;
        #line 11 "impala/gaussian.impala"
        int v_anchor_24622;
        v_anchor_24622 = _24621 / 2;
        #line 60 "impala/gpu_device.impala"
        double* _24660;
        _24660 = _24657 + _24659;
        #line 65 "impala/gpu_device.impala"
        double* _24644;
        union { double* dst; char* src; } u_24644;
        u_24644.src = _24643;
        _24644 = u_24644.dst;
        #line 44 "impala/gaussian.impala"
        bool _24623;
        _24623 = v_anchor_24622 <= gid_y_24617;
        #line 44 "impala/gaussian.impala"
        if (_24623) goto l24624; else goto l24670;
    l24670: ;
        #line 54 "impala/gaussian.impala"
        goto l24664;
    l24624: ;
        #line 44 "impala/gaussian.impala"
        int _24625;
        _24625 = _24618 - v_anchor_24622;
        #line 44 "impala/gaussian.impala"
        bool _24626;
        _24626 = gid_y_24617 < _24625;
        #line 44 "impala/gaussian.impala"
        if (_24626) goto l24627; else goto l24663;
    l24663: ;
        #line 54 "impala/gaussian.impala"
        goto l24664;
    l24664: ;
        #line 65 "impala/gpu_device.impala"
        double* _24665;
        _24665 = _24644 + _24659;
        #line 65 "impala/gpu_device.impala"
        double _24666;
        _24666 = *_24665;
        #line 65 "impala/gpu_device.impala"
        double _24668;
        _24668 = _24666;
        #line 60 "impala/gpu_device.impala"
        *_24660 = _24668;
        return ;
    l24627: ;
        #line 46 "impala/gaussian.impala"
        int _24632;
        _24632 = 1 + v_anchor_24622;
        #line 46 "impala/gaussian.impala"
        int _24662;
        _24662 = 0 - v_anchor_24622;
        #line 70 "impala/gpu_device.impala"
        struct_Buffer_5838 _24636;
        _24636 = _22219_24592.e0;
        #line 70 "impala/gpu_device.impala"
        char* _24637;
        _24637 = _24636.e1;
        #line 70 "impala/gpu_device.impala"
        double* _24638;
        union { double* dst; char* src; } u_24638;
        u_24638.src = _24637;
        _24638 = u_24638.dst;
        #line 19 "impala/gpu_device.impala"
        p_24629 = _24662;
        psum_24631 = 0.000000e+00;
        goto l24628;
    l24628: ;
        _24629 = p_24629;
        sum_24631 = psum_24631;
        #line 19 "impala/gpu_device.impala"
        bool _24633;
        _24633 = _24629 < _24632;
        #line 19 "impala/gpu_device.impala"
        if (_24633) goto l24634; else goto l24655;
    l24655: ;
        #line 60 "impala/gpu_device.impala"
        *_24660 = sum_24631;
        return ;
    l24634: ;
        #line 48 "impala/gaussian.impala"
        int _24639;
        _24639 = _24629 + v_anchor_24622;
        #line 23 "impala/gpu_device.impala"
        int _24635;
        _24635 = 1 + _24629;
        #line 69 "impala/gpu_device.impala"
        double* i_24640;
        i_24640 = _24638 + _24639;
        #line 48 "impala/gaussian.impala"
        int _24645;
        _24645 = gid_y_24617 + _24629;
        #line 70 "impala/gpu_device.impala"
        double _24641;
        _24641 = *i_24640;
        #line 65 "impala/gpu_device.impala"
        int _24646;
        _24646 = _24645 * _24613;
        #line 70 "impala/gpu_device.impala"
        double _24651;
        _24651 = _24641;
        #line 65 "impala/gpu_device.impala"
        int _24647;
        _24647 = _24646 + gid_x_24612;
        #line 65 "impala/gpu_device.impala"
        double* _24648;
        _24648 = _24644 + _24647;
        #line 65 "impala/gpu_device.impala"
        double _24649;
        _24649 = *_24648;
        #line 65 "impala/gpu_device.impala"
        double _24652;
        _24652 = _24649;
        #line 48 "impala/gaussian.impala"
        double _24653;
        _24653 = _24651 * _24652;
        #line 48 "impala/gaussian.impala"
        double _24654;
        _24654 = sum_24631 + _24653;
        #line 19 "impala/gpu_device.impala"
        p_24629 = _24635;
        psum_24631 = _24654;
        goto l24628;
}

}